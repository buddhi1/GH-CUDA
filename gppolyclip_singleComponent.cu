#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <hip/hip_cooperative_groups.h>

#include "constants.h"

typedef struct{
  double x, y;
} point;

__device__ double A(const point& P, const point& Q, const point& R){
	return (Q.x-P.x) * (R.y-P.y) - (Q.y-P.y) * (R.x-P.x);
}

// difference of two 2D points
__device__ point sub(const point& a, const point& b){ 
  point r;
  r.x=a.x-b.x;
  r.y=a.y-b.y;
  return r; 
}

// add two 2D points
__device__ point add(const point& a, const point& b){ 
  point r;
  r.x=a.x+b.x;
  r.y=a.y+b.y;
  return r; 
}

// multiply two 2D points
__device__ double mul(const point& a, const point& b){ 
  point r;
  r.x=a.x*b.x;
  r.y=a.y*b.y;
  return (r.x+r.y); 
}

// multiply scalar with 2D points
__device__ point mulScalar(const double c, const point& b){ 
  point r;
  r.x=c*b.x;
  r.y=c*b.y;
  return r; 
}

/*
-----------------------------------------------------------------
Function to returns the start index of the current id's 
intersections
Returns the intersection starting index
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int getIntersectionStartIndex(int id, int *ps1){
  if(id==0) return 0;
  else return ps1[id];
}

/*
-----------------------------------------------------------------
Function to serach neighbor of a given vertex
Returns the index of the neighbor
Runs in GPU
Called from Device
id: id of the vertex that need to get the neighbor
neighborMap: Own neighborMap. If vertex is from P, Map also from P
neighbor: Other polygon's neighor array. If Id from P, neighbor from Q
-------------------------------------------------------------------
*/
// __device__ int getNeighborIndex(int id, int *neighborMap, int *neighbor){
//   int neighborId=neighborMap[id];
//   if(neighborId!=-100) return neighbor[neighborId];
//   return -1; //no neighbor for this vertex
// }

/*
-----------------------------------------------------------------
Function to return intersection  type
Returns the type of the intersection
Runs in GPU
Called from Device
  NO_INTERSECTION, //0
  X_INTERSECTION,  //1
  T_INTERSECTION_Q, //2
  T_INTERSECTION_P, //3
  V_INTERSECTION, //4
  X_OVERLAP,      //5
  T_OVERLAP_Q,    //6
  T_OVERLAP_P,    //7
  V_OVERLAP       //8
-------------------------------------------------------------------
*/
__device__ int getIntersectType(
            const point& P1, const point& P2, 
            const point& Q1, const point& Q2,  
            double& alpha, double& beta){
	double AP1 = A(P1,Q1,Q2);
	double AP2 = A(P2,Q1,Q2);

	if (fabs(AP1-AP2) > EPSILON){
		// from here: [P1,P2] and [Q1,Q2] are not parallel
		// analyse potential intersection
		double AQ1 = A(Q1,P1,P2);
		double AQ2 = A(Q2,P1,P2);
		// compute alpha and beta
		alpha = AP1 / (AP1-AP2);
		beta  = AQ1 / (AQ1-AQ2);
		// classify alpha
		bool alpha_is_0 = false;
		bool alpha_in_0_1 = false;
		if ( (alpha > EPSILON) && (alpha < 1.0-EPSILON) )
			alpha_in_0_1 = true;
		else
			if (fabs(alpha) <= EPSILON)
				alpha_is_0 = true;
		// classify beta
		bool beta_is_0 = false;
		bool beta_in_0_1 = false;
		if ( (beta > EPSILON) && (beta < 1.0-EPSILON) )
			beta_in_0_1 = true;
		else
			if (fabs(beta) <= EPSILON)
				beta_is_0 = true;
		// distinguish intersection types
		if (alpha_in_0_1 && beta_in_0_1) return (1);  // return (X_INTERSECTION);
		if (alpha_is_0 && beta_in_0_1) return (2);    // return (T_INTERSECTION_Q);
		if (beta_is_0 && alpha_in_0_1) return (3);    // return (T_INTERSECTION_P);
		if (alpha_is_0 && beta_is_0) return (4);      // return (V_INTERSECTION);
	}else if (fabs(AP1) < EPSILON){
			// from here: [P1,P2] and [Q1,Q2] are collinear
			// analyse potential overlap			
      point dP = sub(P2, P1);
			point dQ = sub(Q2, Q1);
			point PQ = sub(Q1, P1);
			alpha = mul(PQ,dP) / mul(dP,dP);
			beta = -mul(PQ,dQ) / mul(dQ,dQ);
			// classify alpha
			bool alpha_is_0 = false;
			bool alpha_in_0_1 = false;
			bool alpha_not_in_0_1 = false;
			if ((alpha > EPSILON) && (alpha < 1.0-EPSILON))
				alpha_in_0_1 = true;
			else
				if (fabs(alpha) <= EPSILON)
					alpha_is_0 = true;
				else
					alpha_not_in_0_1 = true;
			// classify beta
			bool beta_is_0 = false;
			bool beta_in_0_1 = false;
			bool beta_not_in_0_1 = false;
			if ((beta > EPSILON) && (beta < 1.0-EPSILON))
				beta_in_0_1 = true;
			else
				if (fabs(alpha) <= EPSILON)
					beta_is_0 = true;
				else
					beta_not_in_0_1 = true;

			// distinguish intersection types
			if (alpha_in_0_1 && beta_in_0_1) return (5);      // return (X_OVERLAP);
			if (alpha_not_in_0_1 && beta_in_0_1) return (6);  // return (T_OVERLAP_Q);
			if (beta_not_in_0_1 && alpha_in_0_1) return (7);  // return (T_OVERLAP_P);
			if (alpha_is_0 && beta_is_0) return (8);          // return (V_OVERLAP);
		}
  return (0);	// return (NO_INTERSECTION); 
}

/*
-----------------------------------------------------------------
Function to get circular id of a given id 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int getCircularId(int id, int maxCount){
  if(maxCount==id) return 0;
  else if(id==-1) return maxCount-1;
  else return id;
}

/*
-----------------------------------------------------------------
Function to get relative position type
Runs in GPU
Called from Device
0 -> LEFT,
1 -> RIGHT,
2 -> IS_P_m,
3 -> IS_P_p
-------------------------------------------------------------------
*/
__device__ int oracle(int pMNId, int pPNId, int qId, const point& Q, const point& P1, const point& P2, const point& P3) {
  // is Q linked to P1 ?
  if(pMNId!=-100 && pMNId==qId) return 2;
  // is Q linked to P2 ?
  else if(pPNId!=-100 && pPNId==qId) return 3;
  // check relative position of Q with respect to chain (P1,P2,P3)
  double s1 = A(Q, P1, P2);
  double s2 = A(Q, P2, P3);
  double s3 = A(P1, P2, P3);
  if(s3>0){ 
    // chain makes a left turn
    if (s1>0 && s2>0)
      return 0;
    else
      return 1;
  }else{
    // chain makes a right turn (or is straight)
    if(s1<0 && s2<0)
      return 1;
    else
      return 0;
  }
}

/*
-----------------------------------------------------------------
Function to get initial classification label
Runs in GPU
Called from Device
Intersection Labels
0  NONE,
1  CROSSING,
2  BOUNCING,
3  LEFT_ON,
4  RIGHT_ON,
5  ON_ON,
6  ON_LEFT,
7  ON_RIGHT,
8  DELAYED_CROSSING,
9  DELAYED_BOUNCING
-------------------------------------------------------------------
*/
__device__ int getInitialLabel(int qMType, int qPType){
  // check non-overlapping cases
  if((qMType==0  && qPType==1)||(qMType==1 && qPType==0)){
    return 1;
  }
  if((qMType==0  && qPType==0)||(qMType==1 && qPType==1)){
    return 2;
  }
  // check overlapping cases
  if(((qPType==3) && (qMType==1))||((qMType==3) && (qPType==1))) return 3;
  if(((qPType==3) && (qMType==0))||((qMType==3) && (qPType==0))) return 4;
  if(((qPType==3) && (qMType==2))||((qMType==3) && (qPType==2))) return 5;
  if(((qMType==2) && (qPType==1))||((qPType==2) && (qMType==1))) return 6;
  if(((qMType==2) && (qPType==0))||((qPType==2) && (qMType==0))) return 7;
  else return -102;
}

/*
-----------------------------------------------------------------
Function to get a given double value within tolerance 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ double getValueTolarence(double val){
  if(val<EPSILON)
    return 0.0;
  return val;
}

/*
-----------------------------------------------------------------
Function to do counting sort of arr[] according to
  the digit represented by exp.
Returns sorted by single base digit
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ void gpuCountSort(int arr[], int tmpBucket[], int sortedIndicies[], int start, int end, int exp){
  int *output=tmpBucket; // used to track indices w.r.t original araay values
  int i, count[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  // Store count of occurrences in count[]
  for(i=start; i<end; i++){
    *(output+i)=sortedIndicies[i];
    count[(arr[*(output+i)] / exp) % 10]++;
  }
  // count prefix sum contains actual positions
  for(i=1; i<10; i++){
    count[i] += count[i - 1];
  }
  // Build the output array indices
  for(i=end-1; i>=start; i--){
    sortedIndicies[start+(count[(arr[*(output+i)] / exp) % 10]-1)]=*(output+i);
    count[(arr[*(output+i)] / exp) % 10]--;
  }
}

/*
-----------------------------------------------------------------
Function that sorts arr[] of size n using Radix Sort
Returns sorted array
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ void gpuRadixsort(int arr[], int tmpBucket[], int alphaSortedIndicies[], int start, int end){  
  // Do counting sort for every digit. Note that instead
  // of passing digit number, exp is passed. exp is 10^i
  // where i is current digit number
  int i, exp=1;
  for(i=start; i<end; i++){
      alphaSortedIndicies[i]=i;
  }
  for (i=1; i<=EPSILON_POSITIONS; i++){
    gpuCountSort(arr, tmpBucket, alphaSortedIndicies, start, end, exp);
    exp*=10;
  }
  // record sorted alpha values in tmpBucket
  for(i=start; i<end; ++i)
    tmpBucket[i]=arr[alphaSortedIndicies[i]];
}

/*
-----------------------------------------------------------------
Function to return vertex 2 of a given vertex 1
Returns index of vertex 2 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int gpuGetVertex2Index(int vertex1Index, int polySize[], int polyId){
  if(vertex1Index<polySize[polyId]-1) return vertex1Index+1;
  else if(polyId==0) return 0; 
  else return polyId-1;
}

/*
-----------------------------------------------------------------
Function to count all intersections. 
Return prefix sum arrays.
  *prefix sum of count of all intersection vertices x2 (P and Q)
  *prefix sum of count of all intersection vertices excluding 
   degenerate cases x2 (P and Q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCountIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psP2, int *psQ1, int *psQ2){
  int id=blockDim.x*blockIdx.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=sizeQ;
  double *poly1X=polyPX, *poly1Y=polyPY, *poly2X=polyQX, *poly2Y=polyQY;

  if(id>=sizeP+sizeQ) return;
  point P1, P2, Q1, Q2;
  int pid=id;
  if(id>=sizeP){
    size=sizeP;
    poly1X=polyQX; 
    poly1Y=polyQY; 
    poly2X=polyPX;
    poly2Y=polyPY;
    pid=id-sizeP;
  }
  for(int qid=0; qid<size; qid++){
    P1.x = poly1X[pid];
    P1.y = poly1Y[pid];

    Q1.x = poly2X[qid];
    Q1.y = poly2Y[qid];
    Q2.x = poly2X[qid+1];
    Q2.y = poly2Y[qid+1];

    // reset P2 vertex of last edge to first vertex
    if(qid == size-1){
      Q2.x = poly2X[0];
      Q2.y = poly2Y[0];
    }
    //polygon1 is P and polygon2 is Q
    if(pid==id && pid==sizeP-1){
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    }else if(pid!=id && pid == sizeQ-1){ //polygon2 is P and polygon1 is Q
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    } else { //no need reset. Normal case
      P2.x = poly1X[pid+1];
      P2.y = poly1Y[pid+1];
    }

    //
    // determine intersection or overlap type
    //
    // IntersectionType i = intersect(edgeP, edgeQ, alpha, beta);
    int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
    if(i!=0){
      count1++;
      // if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==2 || i==5 || i==6)))
      if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7)))
        count2++;
    }
    // if(id==9 || id==0)
      printf("id %d count1 %d count2  %d P(%f,%f) Q(%f,%f) %d\n", id, count1, count2, P1.x, P1.y, Q1.x, Q1.y, i);

  }
  count2++; //represent the parent vertex 
  if(id<sizeP){
    psP1[pid]=count1;
    psP2[pid]=count2;
  } else{
    psQ1[pid]=count1;
    psQ2[pid]=count2;
  }

  // __syncthreads();
  // thrust::exclusive_scan(thrust::device, psP1, psP1 + sizeP+1, psP1);   //sizeP location contains the total size of the count1
  // thrust::exclusive_scan(thrust::device, psP2, psP2 + sizeP+1, psP2);
  // thrust::exclusive_scan(thrust::device, psQ1, psQ1 + sizeQ+1, psQ1);   //sizeQ location contains the total size of the count1
  // thrust::exclusive_scan(thrust::device, psQ2, psQ2 + sizeQ+1, psQ2);
  // printf("id %d count1 %d count2  %d (%f,%f) (%f,%f)\n", id, count1, count2, P1.x, P1.y, P2.x, P2.y);
  // __syncthreads();
  // if(id==0){
  //   printf("%d \n", sizeP);
  //   for(int ii=0; ii<sizeP; ++ii){
  //     // printf("%d *%d ", psP1[ii], psP2[ii]);
  //     printf("%d ", psP1[ii]);
  //   }
  //   printf("\nend\n");
  //   for(int ii=0; ii<sizeQ; ++ii){
  //     // printf("%d *%d ", psQ1[ii], psQ2[ii]);
  //     printf("%d ", psQ1[ii]);
  //   }
  //   printf("\nend\n");
  // }
}


/*
-----------------------------------------------------------------
Function to neighbor map intersections. 
Return prefix sum arrays.
  *neighbor map all intersection vertices x2 (P and Q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuNeighborMap(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP2, int *psQ2,
                  int *neighborMapP, int *neighborMapQ){
  int id=blockDim.x*blockIdx.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=sizeQ, nonDegenCount=0;
  double *poly1X=polyPX, *poly1Y=polyPY, *poly2X=polyQX, *poly2Y=polyQY;

  if(id>=sizeP+sizeQ) return;
  point P1, P2, Q1, Q2;
  int pid=id;
  if(id>=sizeP){
    size=sizeP;
    poly1X=polyQX; 
    poly1Y=polyQY; 
    poly2X=polyPX;
    poly2Y=polyPY;
    pid=id-sizeP;
    neighborMapQ[psQ2[pid]+count2]=-100;   
  }else{
    neighborMapP[psP2[pid]+count2]=-100;
  }
  for(int qid=0; qid<size; qid++){
    P1.x = poly1X[pid];
    P1.y = poly1Y[pid];

    Q1.x = poly2X[qid];
    Q1.y = poly2Y[qid];
    Q2.x = poly2X[qid+1];
    Q2.y = poly2Y[qid+1];

    // reset P2 vertex of last edge to first vertex
    if(qid == size-1){
      Q2.x = poly2X[0];
      Q2.y = poly2Y[0];
    }
    //polygon1 is P and polygon2 is Q
    if(pid==id && pid==sizeP-1){
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    }else if(pid!=id && pid == sizeQ-1){ //polygon2 is P and polygon1 is Q
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    } else { //no need reset. Normal case
      P2.x = poly1X[pid+1];
      P2.y = poly1Y[pid+1];
    }
    // determine intersection or overlap type
    int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
    if(i!=0){
      count1++;
      if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7))){
        nonDegenCount++;
        count2=nonDegenCount;
      }
      else if((id<sizeP && (i==2 || i==4 || i==6 || i==8)) || (id>=sizeP && (i==2 || i==4 || i==6 || i==8)))
        count2=0;
      if(id<sizeP){
        // if(pid<35) printf("#PPPP# %d %d %d (%f,%f - %f,%f)\n", pid, psP2[pid]+count2, qid, P1.x, P1.y, Q1.x, Q1.y);
        neighborMapP[psP2[pid]+count2]=qid;
      }else{
        // if(pid<35) printf("#qqqq# %d %d %d (%f,%f - %f,%f)\n", pid, psQ2[pid]+count2, qid, P1.x, P1.y, Q1.x, Q1.y);
        neighborMapQ[psQ2[pid]+count2]=qid;
      }
    }
  }
}

/*
-----------------------------------------------------------------
Function to calculate all intersections save them in the correct 
location using prefixsum arrays and make neighbor connections
Returns 
  *intersection arrays with orginal vertices in them x2 (P and Q)
  *neighbor arrays x2 (P and q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCalculateIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX, double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psP2, int *psQ1, int *psQ2, 
                  double *intersectionsP, double *intersectionsQ, double *intersectionsP2, double *intersectionsQ2,
                  int *alphaValuesP, int *alphaValuesQ, int *tmpBucketP, int *tmpBucketQ, int *alphaSortedIndiciesP, int *alphaSortedIndiciesQ,
                  int *neighborP, int *neighborQ, int *neighborP2, int *neighborQ2,
                  int *neighborMapP, int *neighborMapQ, int *neighborMapP2, int *neighborMapQ2,
                  int *initLabelsQ){
  int id=blockDim.x*blockIdx.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, nonDegenCount=0, size=sizeQ, indexIntP, indexIntQ, start, end, localI, neighborQId;
  double *poly1X=polyPX, *poly1Y=polyPY, *poly2X=polyQX, *poly2Y=polyQY;

  if(id>=sizeP+sizeQ) return;

  point P1, P2, Q1, Q2;
  int pid=id;
  if(id>=sizeP){
    size=sizeP;
    poly1X=polyQX; 
    poly1Y=polyQY; 
    poly2X=polyPX;
    poly2Y=polyPY;
    pid=id-sizeP;
    intersectionsQ[psQ2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsQ[psQ2[pid]*2+1]=poly1Y[pid];
    // intersectionsQ[psQ2[pid]*3+2]=-100;     //alpha value define it is a parent, not intersection
    intersectionsQ2[psQ2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsQ2[psQ2[pid]*2+1]=poly1Y[pid];
    alphaValuesQ[psQ2[pid]]=-100;
    // neighborMapQ[psQ2[pid]+count2]=-100;    //default neighbor value. No neighbor
    // neighborMapQ2[psQ2[pid]+count2]=-100;    //default neighbor value. No neighbor
    // printf("id %d loc %d x:%f y:%f\n", id, psQ2[pid], intersectionsQ[psQ2[pid]*3], intersectionsQ[psQ2[pid]*3+1]);
    indexIntQ=getIntersectionStartIndex(pid, psQ1);
  } else {
    intersectionsP[psP2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsP[psP2[pid]*2+1]=poly1Y[pid];
    // intersectionsP[psP2[pid]*3+2]=-100;
    
    intersectionsP2[psP2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsP2[psP2[pid]*2+1]=poly1Y[pid];
    alphaValuesP[psP2[pid]]=-100;
    // neighborMapP[psP2[pid]+count2]=-100;    //default neighbor value. No neighbor
    // neighborMapP2[psP2[pid]+count2]=-100;    //default neighbor value. No neighbor
    // printf("id %d loc %d x:%f y:%f\n", id, psP2[pid], intersectionsP[psP2[pid]*3], intersectionsP[psP2[pid]*3+1]);
    indexIntP=getIntersectionStartIndex(pid, psP1);
  }

  for(int qid=0; qid<size; qid++){
    P1.x = poly1X[pid];
    P1.y = poly1Y[pid];

    Q1.x = poly2X[qid];
    Q1.y = poly2Y[qid];
    Q2.x = poly2X[qid+1];
    Q2.y = poly2Y[qid+1];

    // reset P2 vertex of last edge to first vertex
    if(qid == size-1){
      Q2.x = poly2X[0];
      Q2.y = poly2Y[0];
    }
    //polygon1 is P and polygon2 is Q
    if(pid==id && pid==sizeP-1){
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    }else if(pid!=id && pid == sizeQ-1){ //polygon2 is P and polygon1 is Q
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    } else { //no need reset. Normal case
      P2.x = poly1X[pid+1];
      P2.y = poly1Y[pid+1];
    }

    //
    // determine intersection or overlap type
    //
    // IntersectionType i = intersect(edgeP, edgeQ, alpha, beta);
    int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
    // if(i!=0){
    //   printf("%d %d (%f,%f) %d(%f,%f) %d\n", id, pid, P1.x, P1.y, qid, Q1.x, Q1.y, i);
    // }
    if(i && id<sizeP){
      count1++;
      if(i==1 || i==3 || i==5 || i==7){
        nonDegenCount++;
        count2=nonDegenCount;
      }
      else if(i==2 || i==4 || i==6 || i==8)
        count2=0;
      // neighborMapP[psP2[pid]+count2]=indexIntP+count1-1;
      // neighborMapP2[psP2[pid]+count2]=indexIntP+count1-1;
      // neighborP[indexIntP+count1-1]=psP2[pid]+count2;                    //neighbor of new vertex
      // neighborP2[indexIntP+count1-1]=psP2[pid]+count2; 

      start=psQ2[neighborMapP[psP2[pid]+count2]];
      end=psQ2[neighborMapP[psP2[pid]+count2]+1];
      // printf("***-----***** %d %d %d (%d %d)\n", i, neighborMapP[psP2[pid]+count2], pid, start, end);
      // local search to find the index of qid
      for(localI=start; localI<end; ++localI){
        if(pid==neighborMapQ[localI]){
          neighborQId=localI;
          // if(pid<35) printf("&&& %d %d (%d %d) %d %d\n", id, pid, psP2[pid]+count2, neighborQId, start, neighborMapP[psP2[pid]+count2]);
          neighborP[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
          neighborP2[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
          neighborQ[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
          neighborQ2[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
          localI=end+2; // break;
        }
      }

      switch(i) {
        // case X_INTERSECTION:
        case 1:
          I = add(mulScalar((1.0-alpha), P1), mulScalar(alpha, P2));
          I.x=getValueTolarence(I.x);
          I.y=getValueTolarence(I.y);
          // printf("* %d %d %d %d %f %f\n", (psP2[pid]+count2), indexIntP, count1-1, psP2[pid]+count2, I.x, I.y);
          intersectionsP[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsP[(psP2[pid]+count2)*2+1]=I.y;
          intersectionsP2[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsP2[(psP2[pid]+count2)*2+1]=I.y;
          alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // X-overlap
        case 5:
          // printf("** %d %d %d %d\n", (psP2[pid]+count2), indexIntP, count1-1, psP2[pid]+count2);
          intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
          intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
          alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case T_INTERSECTION_Q:
        // case T_OVERLAP_Q:
        case 2:
        case 6:
          // intersectionsP[psP2[pid]*3+2]=alpha;          //***** error prone. Did not checked in depth
          alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
        break;
        // case T_INTERSECTION_P:
        // case T_OVERLAP_P:
        case 3:
        case 7:
          // printf("*** %d %d %d %d\n", (psP2[pid]+count2), indexIntP, count1-1, psP2[pid]+count2);
          intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
          intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
          alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case V_INTERSECTION:
        // case V_OVERLAP:
        case 4:
        case 8:
          // printf("%d %d (%f,%f) %d(%f,%f) [%d %d %d] %d\n", id, pid, P1.x, P1.y, qid, Q1.x, Q1.y, (psP2[pid]+count2), indexIntP, count1-1, i);
          alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
      } 
    } else if(i && id>=sizeP){
      initLabelsQ[(psQ2[pid]+count2)]=-100;    //make init label to default -100 
      count1++;
      if(i==1 || i==3 || i==5 || i==7){
      // if(i==1 || i==2 || i==5 || i==6){
        nonDegenCount++;
        count2=nonDegenCount;
      }
      else if(i==2 || i==4 || i==6 || i==8)
        count2=0;

        // if(i==1 || i==3 || i==5 || i==7)
        // // if(i==1 || i==2 || i==5 || i==6)
        //   count2++;
        // neighborMapQ[psQ2[pid]+count2]=indexIntQ+count1-1;
        // neighborMapQ2[psQ2[pid]+count2]=indexIntQ+count1-1;
        // neighborQ[indexIntQ+count1-1]=psQ2[pid]+count2;                    //neighbor of new vertex
        // neighborQ2[indexIntQ+count1-1]=psQ2[pid]+count2;  
      switch(i) {
        // case X_INTERSECTION:
        case 1:
          I = add(mulScalar((1.0-alpha), P1), mulScalar(alpha, P2));
          I.x=getValueTolarence(I.x);
          I.y=getValueTolarence(I.y);
          // printf("/* %d %d %d %d %f %f\n", (psQ2[pid]+count2), indexIntQ, count1-1, psQ2[pid]+count2, I.x, I.y);
          intersectionsQ[(psQ2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsQ[(psQ2[pid]+count2)*2+1]=I.y;
          intersectionsQ2[(psQ2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsQ2[(psQ2[pid]+count2)*2+1]=I.y;
          alphaValuesQ[psQ2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case X_OVERLAP:
        case 5:
          // printf("/** %d %d %d %d\n", (psQ2[pid]+count2), indexIntQ, count1-1, psQ2[pid]+count2);
          intersectionsQ[(psQ2[pid]+count2)*2]=Q1.x;    
          intersectionsQ[(psQ2[pid]+count2)*2+1]=Q1.y;
          intersectionsQ2[(psQ2[pid]+count2)*2]=Q1.x;    
          intersectionsQ2[(psQ2[pid]+count2)*2+1]=Q1.y;
          alphaValuesQ[psQ2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*beta;
          break;
        // case T_INTERSECTION_Q:
        // case T_OVERLAP_Q: 
        // was 2, 6
        case 3:
        case 7:
          // printf("/*** %d %d %d %d\n", (psQ2[pid]+count2), indexIntQ, count1-1, psQ2[pid]+count2);
          intersectionsQ[(psQ2[pid]+count2)*2]=Q1.x;
          intersectionsQ[(psQ2[pid]+count2)*2+1]=Q1.y;
          intersectionsQ2[(psQ2[pid]+count2)*2]=Q1.x;
          intersectionsQ2[(psQ2[pid]+count2)*2+1]=Q1.y;
          alphaValuesQ[psQ2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case T_INTERSECTION_P:
        // case T_OVERLAP_P:
        // was 3, 7
        case 2:
        case 6:
          alphaValuesQ[psQ2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
        break;
        // case V_INTERSECTION:
        // case V_OVERLAP:
        case 4:
        case 8:
          // printf("/**** %d %d %d %d\n", (psQ2[pid]+count2), indexIntQ, count1-1, psQ2[pid]+count2);
          // printf("%d %d (%f,%f) %d(%f,%f) [%d %d %d] %d\n", id, pid, P1.x, P1.y, qid, Q1.x, Q1.y, (psQ2[pid]+count2), indexIntQ, count1-1, i);
          alphaValuesQ[psQ2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
      } 
    }
  }
  // --------------------------------------------------------------------------------------------
  // local sort for each edge, start to end
  // --------------------------------------------------------------------------------------------
  if(id<sizeP){
    int start=psP2[pid], end=psP2[pid+1];
    // printf(".. %d %d %d\n", id, start+1, end);
    // for(i=start; i<end; ++i){
    //   printf("(%d %f %f %f %d) \n", id, intersectionsP[i*3], intersectionsP[i*3+1], intersectionsP[i*3+2], alphaValuesP[i]);
    // }
    // sort intersection vertices in this edge locally
    if((end-start)>2){
      gpuRadixsort(alphaValuesP, tmpBucketP, alphaSortedIndiciesP, start+1, end);
      // using sorted index array, change intersection locations in the array and neighbors
      // printf("ssss %d %d %d\n", id, start, end);
      // decending order JUST FOR TESING
      // for(int i=start+1, j=end-1; i<end; ++i, j--){
      // acending order of alpha values 
      for(int i=start+1, j=start+1; i<end; ++i, j++){
        alphaValuesP[i]=tmpBucketP[j];////////////////?????????????????????? need to swap alpha too!!!
        // (x,y,alpha) tuple change in sorted order
        intersectionsP[alphaSortedIndiciesP[j]*2]=intersectionsP2[i*2];
        intersectionsP[alphaSortedIndiciesP[j]*2+1]=intersectionsP2[i*2+1];
        //neighborMap update
        // neighborMapP[alphaSortedIndiciesP[j]]=neighborMapP2[i];
        //neighbor array update
        neighborP[alphaSortedIndiciesP[j]]=neighborP2[i];
        neighborQ[neighborP2[i]-1]=alphaSortedIndiciesP[j]+1; //+1 is the padding. When reading do -1
        neighborQ2[neighborP2[i]-1]=neighborQ[neighborP2[i]-1]; //updates neighborQ2 as the new originla to be used with sorted Q array
      } 
      // for(int i=start, j=end-1; i<end; ++i, --j){
      //   printf("*(%d %d %f %f %d) reverse->%d \n", id, i, intersectionsP[i*2], intersectionsP[i*2+1], alphaValuesP[i], alphaSortedIndiciesP[j]);
      // }
    } 
  }
  // else{
  //   int start=psQ2[pid], end=psQ2[pid+1];
  //   // printf(".. %d %d %d\n", id, start+1, end);
  //   // for(i=start; i<end; ++i){
  //   //   printf("(%d %f %f %f %d) \n", id, intersectionsP[i*3], intersectionsP[i*3+1], intersectionsP[i*3+2], alphaValuesP[i]);
  //   // }
  //   // sort intersection vertices in this edge locally
  //   if((end-start)>2){
  //     gpuRadixsort(alphaValuesQ, tmpBucketQ, alphaSortedIndiciesQ, start+1, end);
  //     // using sorted index array, change intersection locations in the array and neighbors
  //     // printf("ssss %d %d %d\n", id, start, end);
  //     // decending order JUST FOR TESING
  //     // for(int i=start+1, j=end-1; i<end; ++i, j--){
  //     // acending order of alpha values 
  //     for(int i=start+1, j=start+1; i<end; ++i, j++){
  //       alphaValuesQ[i]=tmpBucketQ[j];////////////////?????????????????????? need to swap alpha too!!!
  //       // (x,y,alpha) tuple change in sorted order
  //       intersectionsQ[alphaSortedIndiciesQ[j]*2]=intersectionsQ2[i*2];
  //       intersectionsQ[alphaSortedIndiciesQ[j]*2+1]=intersectionsQ2[i*2+1];
  //       //neighborMap update
  //       // neighborMapQ[alphaSortedIndiciesQ[j]]=neighborMapQ2[i];
  //       //neighbor array update
  //       neighborQ[alphaSortedIndiciesQ[j]]=neighborQ2[i];
  //       neighborP[neighborQ2[i]-1]=alphaSortedIndiciesQ[j]+1; //+1 is the padding. When reading do -1
  //     } 
  //     // for(int i=start, j=end-1; i<end; ++i, --j){
  //     //   printf("****(%d %d %f %f %d) reverse->%d \n", id, i, intersectionsQ[i*2], intersectionsQ[i*2+1], alphaValuesQ[i], alphaSortedIndiciesQ[j]);
  //     // }
  //   } 
  // }
  // --------------------------------------------------------------------------------------------
}

/*
-----------------------------------------------------------------
Function to save vertices of Q in edge wise sorted order
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuSortPolyQ(
                  int sizeP, int sizeQ, 
                  int *psQ2, 
                  double *intersectionsQ, double *intersectionsQ2,
                  int *alphaValuesQ, int *tmpBucketQ,  int *alphaSortedIndiciesQ,
                  int *neighborP, int *neighborQ, int *neighborQ2){
  int id=blockDim.x*blockIdx.x+threadIdx.x;
  if(id>=sizeP && id<(sizeP+sizeQ)){
    int pid=id-sizeP;
    int start=psQ2[pid], end=psQ2[pid+1];
    // sort intersection vertices in this edge locally
    if((end-start)>2){
      gpuRadixsort(alphaValuesQ, tmpBucketQ, alphaSortedIndiciesQ, start+1, end);
      // using sorted index array, change intersection locations in the array and neighbors
      // printf("ssss %d %d %d\n", id, start, end);
      // decending order JUST FOR TESING
      // for(int i=start+1, j=end-1; i<end; ++i, j--){
      // acending order of alpha values 
      for(int i=start+1, j=start+1; i<end; ++i, j++){
        alphaValuesQ[i]=tmpBucketQ[j];////////////////?????????????????????? need to swap alpha too!!!
        // (x,y,alpha) tuple change in sorted order
        intersectionsQ[alphaSortedIndiciesQ[j]*2]=intersectionsQ2[i*2];
        intersectionsQ[alphaSortedIndiciesQ[j]*2+1]=intersectionsQ2[i*2+1];
        //neighbor array update
        neighborQ[alphaSortedIndiciesQ[j]]=neighborQ2[i];
        neighborP[neighborQ2[i]-1]=alphaSortedIndiciesQ[j]+1; //+1 is the padding. When reading do -1
      } 
    } 
  }
}

/*
-----------------------------------------------------------------
Function to calculate initial label
Returns 
  *initial labels x2 (P and Q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCalculateInitLabel(
                int sizeP, int *psP2,
                double *intersectionsP, double *intersectionsQ, int *alphaValuesP, 
                int *neighborP,
                int sizeNP, int sizeNQ, int *initLabelsP, int *initLabelsQ){
  int id=blockDim.x*blockIdx.x+threadIdx.x;
  int pid=id;
  if(id>=sizeP) return;
  int start=psP2[pid], end=psP2[pid+1];
  // int start=psP2[id], end=psP2[id+1];
  int tmpId, nId, pMNId, pPNId;
  point pM, pP, qM, qP, current;
  int qMType, qPType, tmpIniLabel;
  int i;
  for(i=start; i<end; i++){
    initLabelsP[i]=-100;
    // if(intersectionsP[i*2+2]!=-100){    //consider intersections only
    if(alphaValuesP[i]!=-100){    //consider intersections only
      current.x=intersectionsP[i*2]; 
      current.y=intersectionsP[i*2+1]; 
      tmpId=getCircularId(i-1, sizeNP);
      // determine local configuration at this intersection vertex
      pM.x=intersectionsP[tmpId*2];                // P-, predecessor of I on P
      pM.y=intersectionsP[tmpId*2+1];                // P-, predecessor of I on P
      // if(intersectionsP[tmpId*2+2]!=-100)
      if(alphaValuesP[tmpId]!=-100)
        // pMNId=getNeighborIndex(tmpId, neighborMapP, neighborQ); //get neighbor id of P_m vertex
        pMNId=neighborP[tmpId]-1; //get neighbor id of P_m vertex
      else pMNId=-100;

      tmpId=getCircularId(i+1, sizeNP);
      pP.x=intersectionsP[tmpId*2];                // P+, successor of I on P
      pP.y=intersectionsP[tmpId*2+1];                // P+, successor of I on P
      // if(intersectionsP[tmpId*2+2]!=-100)
      if(alphaValuesP[tmpId]!=-100)
        // pPNId=getNeighborIndex(tmpId, neighborMapP, neighborQ); //get neighbor id of P_p vertex
        pPNId=neighborP[tmpId]-1; //get neighbor id of P_p vertex
      else pPNId=-100;

      // nId=getNeighborIndex(i, neighborMapP, neighborQ);
      nId=neighborP[i]-1;
      tmpId=getCircularId(nId-1, sizeNQ);
      qM.x=intersectionsQ[tmpId*2];     // Q-, predecessor of I on Q
      qM.y=intersectionsQ[tmpId*2+1];     // Q-, predecessor of I on Q
      qMType=oracle(pMNId, pPNId, tmpId, qM, pM, current, pP);

      tmpId=getCircularId(nId+1, sizeNQ);
      qP.x=intersectionsQ[tmpId*2];     // Q+, successor of I on P
      qP.y=intersectionsQ[tmpId*2+1];     // Q+, successor of I on P
      qPType=oracle(pMNId, pPNId, tmpId, qP, pM, current, pP);

      tmpIniLabel=getInitialLabel(qMType, qPType);
      initLabelsP[i]=tmpIniLabel;
      initLabelsQ[nId]=tmpIniLabel;
      // printf("%d %d (%f, %f) (%f, %f) (%f, %f) (%f, %f)\n", i, nId, pM.x, pM.y, pP.x, pP.y, qM.x, qM.y, qP.x, qP.y);
      // printf(">>> %d %d %d %d\n", i, qMType, qPType, getInitialLabel(qMType, qPType));
    }
  }
}

/*
-----------------------------------------------------------------
Function to count how many intersection points and prefix sums
Returns 
  *count of non degenerate vertices x2 (P and Q)
  *intersection points with non degenrate vertices included x2
  *neighbor map x2
  *neighbor arrays x2
  *initial labels x2
Neighbor of a vertex (assume index i) in P can be read in O(1) time using
  neighborQ[neighborMapP[i]]
  for Q
    neighborP[neighborMapQ[i]]
Runs in CPU
Called from Host
-------------------------------------------------------------------
*/
void calculateIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *countNonDegenIntP, int *countNonDegenIntQ, 
                  double **intersectionsP, double **intersectionsQ, int **alphaValuesP, int **alphaValuesQ,
                  int **initLabelsP, int **initLabelsQ,
                  int **neighborMapP, int **neighborMapQ, int **neighborP, int **neighborQ){
  double *dev_polyPX, *dev_polyPY, *dev_polyQX, *dev_polyQY;
  int *dev_psP1, *dev_psP2, *dev_psQ1, *dev_psQ2;
  int psP1[sizeP+1], psP2[sizeP+1], psQ1[sizeQ+1], psQ2[sizeQ+1];

  // Phase1: Count intersections in each block. Create prefix sums to find local locations in each thread 
  // Allocate memory in device 
  hipMalloc((void **) &dev_polyPX, sizeP*sizeof(double));
  hipMalloc((void **) &dev_polyPY, sizeP*sizeof(double));
  hipMalloc((void **) &dev_polyQX, sizeQ*sizeof(double));
  hipMalloc((void **) &dev_polyQY, sizeQ*sizeof(double));
  hipMalloc((void **) &dev_psP1, (sizeP+1)*sizeof(int));
  hipMalloc((void **) &dev_psP2, (sizeP+1)*sizeof(int));
  hipMalloc((void **) &dev_psQ1, (sizeQ+1)*sizeof(int));
  hipMalloc((void **) &dev_psQ2, (sizeQ+1)*sizeof(int));

  // Copy input vectors from host memory to GPU buffers.
  hipMemcpy(dev_polyPX, polyPX, sizeP*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_polyPY, polyPY, sizeP*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_polyQX, polyQX, sizeQ*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_polyQY, polyQY, sizeQ*sizeof(double), hipMemcpyHostToDevice);

  // int threadsPerBlock = 2;
  // int blocksPerGrid = (2*(sizeP+sizeQ) + threadsPerBlock - 1) / threadsPerBlock;
  // dim3 dimBlock(threadsPerBlock, 1, 1), dimGrid(blocksPerGrid, 1, 1);   
  int xblocksPerGrid = (2*(sizeP+sizeQ) + xThreadPerBlock - 1) / xThreadPerBlock;
  int yblocksPerGrid = (2*(sizeP+sizeQ) + yThreadPerBlock - 1) / yThreadPerBlock;
  dim3 dimBlock(xThreadPerBlock, yThreadPerBlock, 1), dimGrid(xblocksPerGrid, 1, 1); 
  printf("blockDim %d gridDim %d\n", dimBlock.x, dimGrid.x);

  gpuCountIntersections<<<dimGrid, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ, 
        dev_psP1, dev_psP2, dev_psQ1, dev_psQ2);

  hipMemcpy(&psP1, dev_psP1, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&psP2, dev_psP2, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&psQ1, dev_psQ1, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&psQ2, dev_psQ2, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);

  thrust::exclusive_scan(thrust::host, psP1, psP1 + sizeP+1, psP1);   //sizeP location contains the total size of the count1
  thrust::exclusive_scan(thrust::host, psP2, psP2 + sizeP+1, psP2);
  thrust::exclusive_scan(thrust::host, psQ1, psQ1 + sizeQ+1, psQ1);   //sizeQ location contains the total size of the count1
  thrust::exclusive_scan(thrust::host, psQ2, psQ2 + sizeQ+1, psQ2);

  // for (int i = 0; i < sizeQ+1; ++i){
  // for (int i = 0; i < 15+1; ++i){
  //   printf(" %d-%d ", i, psP2[i]);
  // }
  // printf("--- \n");

  // // for (int i = 0; i < sizeQ+1; ++i){
  // for (int i = 0; i < 15+1; ++i){
  //   printf(" %d-%d ", i, psQ2[i]);
  // }
  // printf("--- \n");
  hipDeviceSynchronize();

  //Phase2: NEW- Fill neighborMap
  int *dev_neighborMapP, *dev_neighborMapQ;
  *countNonDegenIntP=psP2[sizeP];
  *countNonDegenIntQ=psQ2[sizeQ];

  *neighborMapP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *neighborMapQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  
  hipMalloc((void **) &dev_neighborMapP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborMapQ, *countNonDegenIntQ*sizeof(int));

  hipMemcpy(dev_psP1, psP1, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_psP2, psP2, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_psQ1, psQ1, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_psQ2, psQ2, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);

  gpuNeighborMap<<<dimGrid, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ,  
        dev_psP2, dev_psQ2,
        dev_neighborMapP, dev_neighborMapQ);
  
// -----------------------------------------------------------------------------------------------------
  // remove after kernel testing
  // hipMemcpy(*neighborMapP, dev_neighborMapP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(*neighborMapQ, dev_neighborMapQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
// -----------------------------------------------------------------------------------------------------


  // Phase 3: Calcualte intersections and save them in the arrays. Make neighbor connections
  int countIntersections=psP1[sizeP];

  int *alphaSortedIndiciesP, *alphaSortedIndiciesQ;
  double *dev_intersectionsP, *dev_intersectionsQ, *dev_intersectionsP2, *dev_intersectionsQ2;
  int *dev_neighborP, *dev_neighborQ, *dev_neighborP2, *dev_neighborQ2;
  int *dev_neighborMapP2, *dev_neighborMapQ2, *dev_initLabelsP, *dev_initLabelsQ;
  int  *dev_alphaValuesP, *dev_alphaValuesQ, *dev_tmpBucketP, *dev_tmpBucketQ, *dev_alphaSortedIndiciesP, *dev_alphaSortedIndiciesQ;

  *intersectionsP=(double *)malloc(*countNonDegenIntP*2*sizeof(double));
  *intersectionsQ=(double *)malloc(*countNonDegenIntQ*2*sizeof(double));
  *alphaValuesP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *alphaValuesQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  alphaSortedIndiciesP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  alphaSortedIndiciesQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  *initLabelsP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *initLabelsQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  *neighborP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *neighborQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));

  // Allocate memory in device 
  hipMalloc((void **) &dev_intersectionsP, *countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsP2, *countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ, *countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ2, *countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_alphaValuesP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaValuesQ, *countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_tmpBucketP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_tmpBucketQ, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesQ, *countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_neighborP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborP2, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborQ, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_neighborQ2, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_neighborMapP2, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborMapQ2, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_initLabelsQ, *countNonDegenIntQ*sizeof(int));

  gpuCalculateIntersections<<<dimGrid, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ, 
        dev_psP1, dev_psP2, dev_psQ1, dev_psQ2, 
        dev_intersectionsP, dev_intersectionsQ, dev_intersectionsP2, dev_intersectionsQ2,
        dev_alphaValuesP, dev_alphaValuesQ, dev_tmpBucketP, dev_tmpBucketQ, dev_alphaSortedIndiciesP, dev_alphaSortedIndiciesQ,
        dev_neighborP, dev_neighborQ, dev_neighborP2, dev_neighborQ2,
        dev_neighborMapP, dev_neighborMapQ, dev_neighborMapP2, dev_neighborMapQ2,
        dev_initLabelsQ);

  gpuSortPolyQ<<<dimGrid, dimBlock>>>(
        sizeP, sizeQ, 
        dev_psQ2, 
        dev_intersectionsQ, dev_intersectionsQ2,
        dev_alphaValuesQ, dev_tmpBucketQ,  dev_alphaSortedIndiciesQ,
        dev_neighborP, dev_neighborQ, dev_neighborQ2);

  // Phase4: Inital label classificaiton
  // hipMemcpy(*initLabelsQ, dev_initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMalloc((void **) &dev_initLabelsP, *countNonDegenIntP*sizeof(int));
  // hipMemcpy(dev_initLabelsQ, *initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyHostToDevice);
 
  // negative alpha values are not handled explicitly since they are original vertices
  // ******No need to copy alpha values since they are only used to sort edge wise******
  // hipMemcpy(alphaSortedIndicies, dev_alphaSortedIndicies, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);

  gpuCalculateInitLabel<<<dimGrid, dimBlock>>>(
      sizeP,  dev_psP2,
      dev_intersectionsP, dev_intersectionsQ, dev_alphaValuesP,
      dev_neighborP,
      *countNonDegenIntP, *countNonDegenIntQ, dev_initLabelsP, dev_initLabelsQ);

  hipMemcpy(*intersectionsP, dev_intersectionsP, *countNonDegenIntP*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*intersectionsQ, dev_intersectionsQ, *countNonDegenIntQ*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborP, dev_neighborP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborQ, dev_neighborQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborMapP, dev_neighborMapP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborMapQ, dev_neighborMapQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);

  hipMemcpy(*initLabelsP, dev_initLabelsP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*initLabelsQ, dev_initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesP, dev_alphaValuesP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesQ, dev_alphaValuesQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  
  hipDeviceSynchronize();

  // int limitP=*countNonDegenIntP;
  // int limitQ=*countNonDegenIntQ;
  int limitP=10;
  int limitQ=10;

  printf("intersectionP");
  for (int i = 0; i < limitP*2; ++i){
    if(i%2==0) 
      printf("\n%d %d ", i/2, *(*alphaValuesP+(i/2)));
    // printf(" %f ", intersectionsP[i]);
    printf(" %f ", *(*intersectionsP+i));
  }
  printf("\n\nintersectionQ");
  for (int i = 0; i < limitQ*2; ++i){
    if(i%2==0)
      printf("\n%d %d ", i/2, *(*alphaValuesQ+(i/2)));
    printf(" %f ", *(*intersectionsQ+i));
  }
  // printf("\n\nalpha P\n");
  // for (int i = 0; i < *countNonDegenIntP; ++i){
  //   printf(" %d>%d ", i, alphaValuesP[i]);
  // }
  // printf("\n\nalpha Q\n");
  // for (int i = 0; i < *countNonDegenIntQ; ++i){
  //   printf(" %d>%d ", i, alphaValuesQ[i]);
  // }
  // printf("\n");
  printf("\nneighbor P\n");
  for (int i = 0; i < limitP; ++i){
    printf(" %d-%d ", i, *(*neighborP+i));
  }
  printf("\nnneighbor Q\n");
  for (int i = 0; i < limitQ; ++i){
    printf(" %d-%d ", i, *(*neighborQ+i));
  }
  // printf("\n");
  // for (int i = 0; i < *countNonDegenIntP; ++i)
  // {
  //   printf(" %d-%d ", i, *(*neighborMapP+i));
  // }
  // printf("\n");
  // for (int i = 0; i < *countNonDegenIntQ; ++i)
  // {
  //   printf(" %d-%d ", i, *(*neighborMapQ+i));
  // }
  printf("\nLabel P\n");
  for (int i = 0; i < limitP; ++i){
    printf(" %d>%d ", i, *(*initLabelsP+i));
  }
  printf("\nLable Q\n");
  for (int i = 0; i < limitQ; ++i){
    printf(" %d>%d ", i, *(*initLabelsQ+i));
  }
  printf("\n");


  hipFree(dev_polyPX);
  hipFree(dev_polyPY);
  hipFree(dev_polyQX);
  hipFree(dev_polyQY);
}

/*
-----------------------------------------------------------------
Function to count how many intersection points and prefix sums
  Works with multiple components in PP and QQ
Returns 
  *count of non degenerate vertices x2 (P and Q)
  *intersection points with non degenrate vertices included x2
  *neighbor map x2
  *neighbor arrays x2
  *initial labels x2
Neighbor of a vertex (assume index i) in P can be read in O(1) time using
  neighborQ[neighborMapP[i]]
  for Q
    neighborP[neighborMapQ[i]]
Runs in CPU
Called from Host
-------------------------------------------------------------------
*/
void calculateIntersectionsMultipleComponents(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, int *sizesPP, int *sizesQQ, int sizePP, int sizeQQ,
                  int *countNonDegenIntArrayP, int *countNonDegenIntArrayQ, 
                  double **intersectionsP, double **intersectionsQ, int **alphaValuesP, int **alphaValuesQ,
                  int **initLabelsP, int **initLabelsQ,
                  int **neighborMapP, int **neighborMapQ, int **neighborP, int **neighborQ){
  double *dev_polyPX, *dev_polyPY, *dev_polyQX, *dev_polyQY;
  int *dev_psP1, *dev_psP2, *dev_psQ1, *dev_psQ2;
  int psP1[sizeP+1], psP2[sizeP+1], psQ1[sizeQ+1], psQ2[sizeQ+1];

  
  printf("\nP ");
  for (int i = 0; i < sizeP; ++i){
  // for (int i = 0; i < 15+1; ++i){
    printf(" %d-%f,%f \n", i, polyPX[i], polyPY[i]);
  }
  printf("--- \nQ ");
  for (int i = 0; i < sizeQ; ++i){
  // for (int i = 0; i < 15+1; ++i){
    printf(" %d-%f,%f\n ", i, polyQX[i], polyQY[i]);
  }
  printf("\n");
  // Phase1: Count intersections in each block. Create prefix sums to find local locations in each thread 
  // Allocate memory in device 
  hipMalloc((void **) &dev_polyPX, sizeP*sizeof(double));
  hipMalloc((void **) &dev_polyPY, sizeP*sizeof(double));
  hipMalloc((void **) &dev_polyQX, sizeQ*sizeof(double));
  hipMalloc((void **) &dev_polyQY, sizeQ*sizeof(double));
  hipMalloc((void **) &dev_psP1, (sizeP+1)*sizeof(int));
  hipMalloc((void **) &dev_psP2, (sizeP+1)*sizeof(int));
  hipMalloc((void **) &dev_psQ1, (sizeQ+1)*sizeof(int));
  hipMalloc((void **) &dev_psQ2, (sizeQ+1)*sizeof(int));

  // Copy input vectors from host memory to GPU buffers.
  hipMemcpy(dev_polyPX, polyPX, sizeP*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_polyPY, polyPY, sizeP*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_polyQX, polyQX, sizeQ*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_polyQY, polyQY, sizeQ*sizeof(double), hipMemcpyHostToDevice);

  // int threadsPerBlock = 2;
  // int blocksPerGrid = (2*(sizeP+sizeQ) + threadsPerBlock - 1) / threadsPerBlock;
  // dim3 dimBlock(threadsPerBlock, 1, 1), dimGrid(blocksPerGrid, 1, 1);   
  int xblocksPerGrid = (2*(sizeP+sizeQ) + xThreadPerBlock - 1) / xThreadPerBlock;
  int yblocksPerGrid = (2*(sizeP+sizeQ) + yThreadPerBlock - 1) / yThreadPerBlock;
  dim3 dimBlock(xThreadPerBlock, yThreadPerBlock, 1), dimGrid(xblocksPerGrid, 1, 1); 
  printf("blockDim %d gridDim %d\n", dimBlock.x, dimGrid.x);

  gpuCountIntersections<<<dimGrid, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ, 
        dev_psP1, dev_psP2, dev_psQ1, dev_psQ2);

  hipMemcpy(&psP1, dev_psP1, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&psP2, dev_psP2, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&psQ1, dev_psQ1, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&psQ2, dev_psQ2, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);

  thrust::exclusive_scan(thrust::host, psP1, psP1 + sizeP+1, psP1);   //sizeP location contains the total size of the count1
  thrust::exclusive_scan(thrust::host, psP2, psP2 + sizeP+1, psP2);
  thrust::exclusive_scan(thrust::host, psQ1, psQ1 + sizeQ+1, psQ1);   //sizeQ location contains the total size of the count1
  thrust::exclusive_scan(thrust::host, psQ2, psQ2 + sizeQ+1, psQ2);

  printf("\nsizesPP ");
  for (int i = 0; i < sizePP; ++i){
  // for (int i = 0; i < 15+1; ++i){
    printf(" %d-%d ", i, sizesPP[i]);
  }
  printf("--- \nsizesQQ ");
  for (int i = 0; i < sizeQQ; ++i){
  // for (int i = 0; i < 15+1; ++i){
    printf(" %d-%d ", i, sizesQQ[i]);
  }
  printf("--- \npsP2 ");
  for (int i = 0; i < sizeP+1; ++i){
  // for (int i = 0; i < 15+1; ++i){
    printf(" %d-%d ", i, psP2[i]);
  }
  printf("--- \npsQ2 ");

  for (int i = 0; i < sizeQ+1; ++i){
  // for (int i = 0; i < 15+1; ++i){
    printf(" %d-%d ", i, psQ2[i]);
  }
  printf("--- \n");
  hipDeviceSynchronize();

  //Phase2: NEW- Fill neighborMap
  int *dev_neighborMapP, *dev_neighborMapQ;
  int countNonDegenIntP, countNonDegenIntQ;
  int count=0;
  for(int i=0; i<sizePP; ++i){
    count+=sizesPP[i];
    countNonDegenIntArrayP[i]=psP2[count];
  }
  count=0;
  for(int i=0; i<sizeQQ; ++i){
    count+=sizesQQ[i];
    countNonDegenIntArrayQ[i]=psQ2[count];
  }
  countNonDegenIntP=psP2[sizeP];
  countNonDegenIntQ=psQ2[sizeQ];

  *neighborMapP=(int *)malloc(countNonDegenIntP*sizeof(int));
  *neighborMapQ=(int *)malloc(countNonDegenIntQ*sizeof(int));
  
  hipMalloc((void **) &dev_neighborMapP, countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborMapQ, countNonDegenIntQ*sizeof(int));

  hipMemcpy(dev_psP1, psP1, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_psP2, psP2, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_psQ1, psQ1, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_psQ2, psQ2, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);

  gpuNeighborMap<<<dimGrid, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ,  
        dev_psP2, dev_psQ2,
        dev_neighborMapP, dev_neighborMapQ);
  
// -----------------------------------------------------------------------------------------------------
  // remove after kernel testing
  // hipMemcpy(*neighborMapP, dev_neighborMapP, countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(*neighborMapQ, dev_neighborMapQ, countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
// -----------------------------------------------------------------------------------------------------


  // Phase 3: Calcualte intersections and save them in the arrays. Make neighbor connections
  int countIntersections=psP1[sizeP];

  int *alphaSortedIndiciesP, *alphaSortedIndiciesQ;
  double *dev_intersectionsP, *dev_intersectionsQ, *dev_intersectionsP2, *dev_intersectionsQ2;
  int *dev_neighborP, *dev_neighborQ, *dev_neighborP2, *dev_neighborQ2;
  int *dev_neighborMapP2, *dev_neighborMapQ2, *dev_initLabelsP, *dev_initLabelsQ;
  int  *dev_alphaValuesP, *dev_alphaValuesQ, *dev_tmpBucketP, *dev_tmpBucketQ, *dev_alphaSortedIndiciesP, *dev_alphaSortedIndiciesQ;

  *intersectionsP=(double *)malloc(countNonDegenIntP*2*sizeof(double));
  *intersectionsQ=(double *)malloc(countNonDegenIntQ*2*sizeof(double));
  *alphaValuesP=(int *)malloc(countNonDegenIntP*sizeof(int));
  *alphaValuesQ=(int *)malloc(countNonDegenIntQ*sizeof(int));
  alphaSortedIndiciesP=(int *)malloc(countNonDegenIntP*sizeof(int));
  alphaSortedIndiciesQ=(int *)malloc(countNonDegenIntQ*sizeof(int));
  *initLabelsP=(int *)malloc(countNonDegenIntP*sizeof(int));
  *initLabelsQ=(int *)malloc(countNonDegenIntQ*sizeof(int));
  *neighborP=(int *)malloc(countNonDegenIntP*sizeof(int));
  *neighborQ=(int *)malloc(countNonDegenIntQ*sizeof(int));

  // Allocate memory in device 
  hipMalloc((void **) &dev_intersectionsP, countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsP2, countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ, countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ2, countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_alphaValuesP, countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaValuesQ, countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_tmpBucketP, countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_tmpBucketQ, countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesP, countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesQ, countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_neighborP, countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborP2, countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborQ, countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_neighborQ2, countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_neighborMapP2, countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborMapQ2, countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_initLabelsQ, countNonDegenIntQ*sizeof(int));

  gpuCalculateIntersections<<<dimGrid, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ, 
        dev_psP1, dev_psP2, dev_psQ1, dev_psQ2, 
        dev_intersectionsP, dev_intersectionsQ, dev_intersectionsP2, dev_intersectionsQ2,
        dev_alphaValuesP, dev_alphaValuesQ, dev_tmpBucketP, dev_tmpBucketQ, dev_alphaSortedIndiciesP, dev_alphaSortedIndiciesQ,
        dev_neighborP, dev_neighborQ, dev_neighborP2, dev_neighborQ2,
        dev_neighborMapP, dev_neighborMapQ, dev_neighborMapP2, dev_neighborMapQ2,
        dev_initLabelsQ);

  gpuSortPolyQ<<<dimGrid, dimBlock>>>(
        sizeP, sizeQ, 
        dev_psQ2, 
        dev_intersectionsQ, dev_intersectionsQ2,
        dev_alphaValuesQ, dev_tmpBucketQ,  dev_alphaSortedIndiciesQ,
        dev_neighborP, dev_neighborQ, dev_neighborQ2);

  // Phase4: Inital label classificaiton
  // hipMemcpy(*initLabelsQ, dev_initLabelsQ, countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMalloc((void **) &dev_initLabelsP, countNonDegenIntP*sizeof(int));
  // hipMemcpy(dev_initLabelsQ, *initLabelsQ, countNonDegenIntQ*sizeof(int), hipMemcpyHostToDevice);
 
  // negative alpha values are not handled explicitly since they are original vertices
  // ******No need to copy alpha values since they are only used to sort edge wise******
  // hipMemcpy(alphaSortedIndicies, dev_alphaSortedIndicies, countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);

  gpuCalculateInitLabel<<<dimGrid, dimBlock>>>(
      sizeP,  dev_psP2,
      dev_intersectionsP, dev_intersectionsQ, dev_alphaValuesP,
      dev_neighborP,
      countNonDegenIntP, countNonDegenIntQ, dev_initLabelsP, dev_initLabelsQ);

  hipMemcpy(*intersectionsP, dev_intersectionsP, countNonDegenIntP*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*intersectionsQ, dev_intersectionsQ, countNonDegenIntQ*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborP, dev_neighborP, countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborQ, dev_neighborQ, countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborMapP, dev_neighborMapP, countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborMapQ, dev_neighborMapQ, countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);

  hipMemcpy(*initLabelsP, dev_initLabelsP, countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*initLabelsQ, dev_initLabelsQ, countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesP, dev_alphaValuesP, countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesQ, dev_alphaValuesQ, countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  
  hipDeviceSynchronize();

  // int limitP=countNonDegenIntP;
  // int limitQ=countNonDegenIntQ;
  int limitP=10;
  int limitQ=10;

  printf("intersectionP");
  for (int i = 0; i < limitP*2; ++i){
    if(i%2==0) 
      printf("\n%d %d ", i/2, *(*alphaValuesP+(i/2)));
    // printf(" %f ", intersectionsP[i]);
    printf(" %f ", *(*intersectionsP+i));
  }
  printf("\n\nintersectionQ");
  for (int i = 0; i < limitQ*2; ++i){
    if(i%2==0)
      printf("\n%d %d ", i/2, *(*alphaValuesQ+(i/2)));
    printf(" %f ", *(*intersectionsQ+i));
  }
  // printf("\n\nalpha P\n");
  // for (int i = 0; i < limitP; ++i){
  //   printf(" %d>%d ", i, alphaValuesP[i]);
  // }
  // printf("\n\nalpha Q\n");
  // for (int i = 0; i < limitQ; ++i){
  //   printf(" %d>%d ", i, alphaValuesQ[i]);
  // }
  // printf("\n");
  printf("\nneighbor P\n");
  for (int i = 0; i < limitP; ++i){
    printf(" %d-%d ", i, *(*neighborP+i));
  }
  printf("\nneighbor Q\n");
  for (int i = 0; i < limitQ; ++i){
    printf(" %d-%d ", i, *(*neighborQ+i));
  }
  // printf("\n");
  // for (int i = 0; i < limitP; ++i)
  // {
  //   printf(" %d-%d ", i, *(*neighborMapP+i));
  // }
  // printf("\n");
  // for (int i = 0; i < limitQ; ++i)
  // {
  //   printf(" %d-%d ", i, *(*neighborMapQ+i));
  // }
  printf("\nLabel P\n");
  for (int i = 0; i < limitP; ++i){
    printf(" %d>%d ", i, *(*initLabelsP+i));
  }
  printf("\nLable Q\n");
  for (int i = 0; i < limitQ; ++i){
    printf(" %d>%d ", i, *(*initLabelsQ+i));
  }
  printf("\n");


  hipFree(dev_polyPX);
  hipFree(dev_polyPY);
  hipFree(dev_polyQX);
  hipFree(dev_polyQY);
}