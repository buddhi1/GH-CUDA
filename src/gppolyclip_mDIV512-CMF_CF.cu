#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <hip/hip_cooperative_groups.h>

#include "../lib/constants.h"

typedef struct{
  double x, y;
} point;

__device__ double A(const point& P, const point& Q, const point& R){
	return (Q.x-P.x) * (R.y-P.y) - (Q.y-P.y) * (R.x-P.x);
}

// difference of two 2D points
__device__ point sub(const point& a, const point& b){ 
  point r;
  r.x=a.x-b.x;
  r.y=a.y-b.y;
  return r; 
}

// add two 2D points
__device__ point add(const point& a, const point& b){ 
  point r;
  r.x=a.x+b.x;
  r.y=a.y+b.y;
  return r; 
}

// multiply two 2D points
__device__ double mul(const point& a, const point& b){ 
  point r;
  r.x=a.x*b.x;
  r.y=a.y*b.y;
  return (r.x+r.y); 
}

// multiply scalar with 2D points
__device__ point mulScalar(const double c, const point& b){ 
  point r;
  r.x=c*b.x;
  r.y=c*b.y;
  return r; 
}

// find min
__device__ double getMin(double a, double b){
  if(a<b) return a;
  return b;
}

// find max
__device__ double getMax(double a, double b){
  if(a<b) return b;
  return a;
}

/*
-----------------------------------------------------------------
Function to returns the start index of the current id's 
intersections
Returns the intersection starting index
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int getIntersectionStartIndex(int id, int *ps1){
  if(id==0) return 0;
  else return ps1[id];
}

/*
-----------------------------------------------------------------
Function to return intersection  type
Returns the type of the intersection
Runs in GPU
Called from Device
  NO_INTERSECTION, //0
  X_INTERSECTION,  //1
  T_INTERSECTION_Q, //2
  T_INTERSECTION_P, //3
  V_INTERSECTION, //4
  X_OVERLAP,      //5
  T_OVERLAP_Q,    //6
  T_OVERLAP_P,    //7
  V_OVERLAP       //8
-------------------------------------------------------------------
*/
__device__ int getIntersectType(
            const point& P1, const point& P2, 
            const point& Q1, const point& Q2,  
            double& alpha, double& beta){
	double AP1 = A(P1,Q1,Q2);
	double AP2 = A(P2,Q1,Q2);

	if (fabs(AP1-AP2) > EPSILON){
		// from here: [P1,P2] and [Q1,Q2] are not parallel
		// analyse potential intersection
		double AQ1 = A(Q1,P1,P2);
		double AQ2 = A(Q2,P1,P2);
		// compute alpha and beta
		alpha = AP1 / (AP1-AP2);
		beta  = AQ1 / (AQ1-AQ2);
		// classify alpha
		bool alpha_is_0 = false;
		bool alpha_in_0_1 = false;
		if ( (alpha > EPSILON) && (alpha < 1.0-EPSILON) )
			alpha_in_0_1 = true;
		else
			if (fabs(alpha) <= EPSILON)
				alpha_is_0 = true;
		// classify beta
		bool beta_is_0 = false;
		bool beta_in_0_1 = false;
		if ( (beta > EPSILON) && (beta < 1.0-EPSILON) )
			beta_in_0_1 = true;
		else
			if (fabs(beta) <= EPSILON)
				beta_is_0 = true;
		// distinguish intersection types
		if (alpha_in_0_1 && beta_in_0_1) return (1);  // return (X_INTERSECTION);
		if (alpha_is_0 && beta_in_0_1) return (2);    // return (T_INTERSECTION_Q);
		if (beta_is_0 && alpha_in_0_1) return (3);    // return (T_INTERSECTION_P);
		if (alpha_is_0 && beta_is_0) return (4);      // return (V_INTERSECTION);
	}else if (fabs(AP1) < EPSILON){
			// from here: [P1,P2] and [Q1,Q2] are collinear
			// analyse potential overlap			
      point dP = sub(P2, P1);
			point dQ = sub(Q2, Q1);
			point PQ = sub(Q1, P1);
			alpha = mul(PQ,dP) / mul(dP,dP);
			beta = -mul(PQ,dQ) / mul(dQ,dQ);
			// classify alpha
			bool alpha_is_0 = false;
			bool alpha_in_0_1 = false;
			bool alpha_not_in_0_1 = false;
			if ((alpha > EPSILON) && (alpha < 1.0-EPSILON))
				alpha_in_0_1 = true;
			else
				if (fabs(alpha) <= EPSILON)
					alpha_is_0 = true;
				else
					alpha_not_in_0_1 = true;
			// classify beta
			bool beta_is_0 = false;
			bool beta_in_0_1 = false;
			bool beta_not_in_0_1 = false;
			if ((beta > EPSILON) && (beta < 1.0-EPSILON))
				beta_in_0_1 = true;
			else
				if (fabs(alpha) <= EPSILON)
					beta_is_0 = true;
				else
					beta_not_in_0_1 = true;

			// distinguish intersection types
			if (alpha_in_0_1 && beta_in_0_1) return (5);      // return (X_OVERLAP);
			if (alpha_not_in_0_1 && beta_in_0_1) return (6);  // return (T_OVERLAP_Q);
			if (beta_not_in_0_1 && alpha_in_0_1) return (7);  // return (T_OVERLAP_P);
			if (alpha_is_0 && beta_is_0) return (8);          // return (V_OVERLAP);
		}
  return (0);	// return (NO_INTERSECTION); 
}

/*
-----------------------------------------------------------------
Function to get circular id of a given id 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int getCircularId(int id, int maxCount){
  if(maxCount==id) return 0;
  else if(id==-1) return maxCount-1;
  else return id;
}

/*
-----------------------------------------------------------------
Function to get relative position type
Runs in GPU
Called from Device
0 -> LEFT,
1 -> RIGHT,
2 -> IS_P_m,
3 -> IS_P_p
-------------------------------------------------------------------
*/
__device__ int oracle(int pMNId, int pPNId, int qId, const point& Q, const point& P1, const point& P2, const point& P3) {
  // is Q linked to P1 ?
  if(pMNId!=-100 && pMNId==qId) return 2;
  // is Q linked to P2 ?
  else if(pPNId!=-100 && pPNId==qId) return 3;
  // check relative position of Q with respect to chain (P1,P2,P3)
  double s1 = A(Q, P1, P2);
  double s2 = A(Q, P2, P3);
  double s3 = A(P1, P2, P3);
  if(s3>0){ 
    // chain makes a left turn
    if (s1>0 && s2>0)
      return 0;
    else
      return 1;
  }else{
    // chain makes a right turn (or is straight)
    if(s1<0 && s2<0)
      return 1;
    else
      return 0;
  }
}

/*
-----------------------------------------------------------------
Function to get initial classification label
Runs in GPU
Called from Device
Intersection Labels
0  NONE,
1  CROSSING,
2  BOUNCING,
3  LEFT_ON,
4  RIGHT_ON,
5  ON_ON,
6  ON_LEFT,
7  ON_RIGHT,
8  DELAYED_CROSSING,
9  DELAYED_BOUNCING
-------------------------------------------------------------------
*/
__device__ int getInitialLabel(int qMType, int qPType){
  // check non-overlapping cases
  if((qMType==0  && qPType==1)||(qMType==1 && qPType==0)){
    return 1;
  }
  if((qMType==0  && qPType==0)||(qMType==1 && qPType==1)){
    return 2;
  }
  // check overlapping cases
  if(((qPType==3) && (qMType==1))||((qMType==3) && (qPType==1))) return 3;
  if(((qPType==3) && (qMType==0))||((qMType==3) && (qPType==0))) return 4;
  if(((qPType==3) && (qMType==2))||((qMType==3) && (qPType==2))) return 5;
  if(((qMType==2) && (qPType==1))||((qPType==2) && (qMType==1))) return 6;
  if(((qMType==2) && (qPType==0))||((qPType==2) && (qMType==0))) return 7;
  else return -102;
}

/*
-----------------------------------------------------------------
Function to do counting sort of arr[] according to
  the digit represented by exp.
Returns sorted by single base digit
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ void gpuCountSort(int arr[], int tmpBucket[], int sortedIndicies[], int start, int end, int exp){
  int *output=tmpBucket; // used to track indices w.r.t original araay values
  int i, count[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  // Store count of occurrences in count[]
  for(i=start; i<end; i++){
    *(output+i)=sortedIndicies[i];
    count[(arr[*(output+i)] / exp) % 10]++;
  }
  // count prefix sum contains actual positions
  for(i=1; i<10; i++){
    count[i] += count[i - 1];
  }
  // Build the output array indices
  for(i=end-1; i>=start; i--){
    sortedIndicies[start+(count[(arr[*(output+i)] / exp) % 10]-1)]=*(output+i);
    count[(arr[*(output+i)] / exp) % 10]--;
  }
}

/*
-----------------------------------------------------------------
Function that sorts arr[] of size n using Radix Sort
Returns sorted array
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ void gpuRadixsort(int arr[], int tmpBucket[], int alphaSortedIndicies[], int start, int end){  
  // Do counting sort for every digit. Note that instead
  // of passing digit number, exp is passed. exp is 10^i
  // where i is current digit number
  int i, exp=1;
  for(i=start; i<end; i++){
      alphaSortedIndicies[i]=i;
  }
  for (i=1; i<=EPSILON_POSITIONS; i++){
    gpuCountSort(arr, tmpBucket, alphaSortedIndicies, start, end, exp);
    exp*=10;
  }
  // record sorted alpha values in tmpBucket
  for(i=start; i<end; ++i)
    tmpBucket[i]=arr[alphaSortedIndicies[i]];
}

/*
-----------------------------------------------------------------
Function to return vertex 2 of a given vertex 1
Returns index of vertex 2 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int gpuGetVertex2Index(int vertex1Index, int polySize[], int polyId){
  if(vertex1Index<polySize[polyId+1]-1) return vertex1Index+1;
  else if(vertex1Index=polySize[polyId+1]-1) return polySize[polyId];
}

/*
-----------------------------------------------------------------
Function: iterative search 
Returns location of x in given array arr[l..r] if present,
  otherwise -1
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int gpuSearchPolygonId(int arr[], int numPol, int x){
  for(int i=0; i<numPol; ++i){
    if(arr[i]<=x && arr[i+1]>x)
      return i;
  }
  return -1;
}

/*
-----------------------------------------------------------------
Function to check if there is a overlap between given 2 edges 
Returns 1 if there is a overlap; else 0
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int gpuLSMF(point P1, point P2, point Q1, point Q2){
  double minPX=P1.x, minPY=P1.y;
  double maxPX=P2.x, maxPY=P2.y;
  double minQX=Q1.x, minQY=Q1.y;
  double maxQX=Q2.x, maxQY=Q2.y;
  // this toggle way optimizes this computation well compared to using 8 min max calls seperately
  if(minPX>P2.x){
    minPX=P2.x;
    maxPX=P1.x;
  }
  if(minPY>P2.y){
    minPY=P2.y;
    maxPY=P1.y;
  }
  if(minQX>Q2.x){
    minQX=Q2.x;
    maxQX=Q1.x;
  }
  if(minQY>Q2.y){
    minQY=Q2.y;
    maxQY=Q1.y;
  }
  // check intersection between MBRs
  if(minPX>maxQX || maxPX<minQX) return 0;
  if(minPY>maxQY || maxPY<minQY) return 0;
  return 1;
}

/*
-----------------------------------------------------------------
Function to check if edegs are intersecting with the CMBR
Return prefix sum arrays.
  if a marked boolean array if the edges are intersecting with it
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCMBRFilter(
                double *polyX, double *polyY, 
                double cmbrMinX, double cmbrMinY, double cmbrMaxX, double cmbrMaxY,
                int size, int *boolPs, int *ps1, int *ps2){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  
  if(id>size) return;
  
  point P1, P2;
  P1.x=polyX[id];
  P1.y=polyY[id];
  P2.x=polyX[(id+1)%size];
  P2.y=polyY[(id+1)%size];

  double minX=getMin(P1.x, P2.x), minY=getMin(P1.y, P2.y);
  double maxX=getMax(P1.x, P2.x), maxY=getMax(P1.y, P2.y);

  boolPs[id]=1;
  ps1[id]=0;
  ps2[id]=1; //by default paren is in the list. Hence the initial value
  if(minX>cmbrMaxX || maxX<cmbrMinX) boolPs[id]=0;
  if(minY>cmbrMaxY || maxY<cmbrMinY) boolPs[id]=0;
  // if(boolPs[id]!=1) printf("/// %d\n", id);
}

/*
-----------------------------------------------------------------
Function to record all indicies which intersects with CMBR 
Return prefix sum arrays.
  index arrays
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuSaveCMBRIntersectedIndicies(
                double *polyX, double *polyY, 
                double cmbrMinX, double cmbrMinY, double cmbrMaxX, double cmbrMaxY,
                int size, int *boolPol, int *boolPs){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  
  if(id>size) return;
  
  point P1, P2;
  P1.x=polyX[id];
  P1.y=polyY[id];
  P2.x=polyX[(id+1)%size];
  P2.y=polyY[(id+1)%size];

  double minX=getMin(P1.x, P2.x), minY=getMin(P1.y, P2.y);
  double maxX=getMax(P1.x, P2.x), maxY=getMax(P1.y, P2.y);

  int intersect=1;
  if(minX>cmbrMaxX || maxX<cmbrMinX) intersect=0;
  if(minY>cmbrMaxY || maxY<cmbrMinY) intersect=0;
  if(intersect){
    boolPol[boolPs[id]]=id;
    // if(boolPs[id]!=id) printf("Error %d %d \n", id, boolPs[id]);
  }
}

/*
-----------------------------------------------------------------
Function to count all intersections. Simple bool check CMBR filter
Return prefix sum arrays.
  *prefix sum of count of all intersection vertices x2 (P and Q)
  *prefix sum of count of all intersection vertices excluding 
   degenerate cases x2 (P and Q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCountIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ,
                  int *psP1, int *psP2, int *boolPIndex){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  int idx=threadIdx.x;  
  __shared__ double poly2X_shared[MAX_POLY2_SIZE+1], poly2Y_shared[MAX_POLY2_SIZE+1] /*+1 for halo next*/;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=0, qid;
  point P1, P2, Q1, Q2;

  int tiles=(sizeQ+MAX_POLY2_SIZE-1)/MAX_POLY2_SIZE;
  int tileCellsPerThread=MAX_POLY2_SIZE/blockDim.x;
  if(id<sizeP){
    P1.x = polyPX[id];
    P1.y = polyPY[id];
    P2.x = polyPX[(id+1)%sizeP];
    P2.y = polyPY[(id+1)%sizeP];
  }
  for(int tileId=0; tileId<tiles; tileId++){
    size=MAX_POLY2_SIZE;
    qid=idx*SHARED_MEMORY_PADDING;
    if(tileId==tiles-1 && sizeQ%MAX_POLY2_SIZE!=0){
      size=sizeQ%MAX_POLY2_SIZE;
      qid=0;
    }
    for(int localId=0; localId<tileCellsPerThread; ++localId){
      if(tileId!=tiles-1 || (tileId==tiles-1 && idx<size)){
        // load data into shared memory collaboratively
        poly2X_shared[idx+(blockDim.x*localId)]=polyQX[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)];
        poly2Y_shared[idx+(blockDim.x*localId)]=polyQY[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)];
        if(tileId!=tiles-1 && idx==blockDim.x-1 && localId==tileCellsPerThread-1){
          poly2X_shared[idx+(blockDim.x*localId)+1]=polyQX[idx+(blockDim.x*localId)+1+(tileId*MAX_POLY2_SIZE)];
          poly2Y_shared[idx+(blockDim.x*localId)+1]=polyQY[idx+(blockDim.x*localId)+1+(tileId*MAX_POLY2_SIZE)];
        }
      }
    } 
    __syncthreads();
    if(boolPIndex[id]) 
    {
      for(int qCount=0; qCount<size; qid=((qid+1)%size), ++qCount){   
      // for(int qid=0; qid<size; qid++){  
        Q1.x = poly2X_shared[qid];
        Q1.y = poly2Y_shared[qid];
        // reset P2 vertex of last edge to first vertex
        if(tileId==tiles-1 && qid==size-1){
          Q2.x=polyQX[0];
          Q2.y=polyQY[0];
        }else{
          Q2.x=poly2X_shared[qid+1];
          Q2.y=poly2Y_shared[qid+1];
        }      
        // if MBRs of two edges does not have a CMBR, there cannot be any intersection at all
        // if(gpuLSMF(P1, P2, Q1, Q2))
        {
          // determine intersection or overlap type
          int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
          if(i!=0){
            count1++;
            if(i==1 || i==3 || i==5 || i==7)
              count2++;
          }        
        }
      } 
    }
    __syncthreads();
  }
  if(id<sizeP){
    count2++; //represent the parent vertex 
    psP1[id]=count1;
    psP2[id]=count2; 
  }
}
__global__ void gpuNeighborMap(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psQ1, int *psQ2,
                  int *neighborMapQ){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, nonDegenCount=0;

  if(id>=sizeQ) return;

  neighborMapQ[psQ2[id]+count2]=-100;  
  // check if the current edge has any intersections. If not return
  // printf("id %d %d %d \n", id, psQ1[id], psQ1[id+1]);
  // CMBR filter: check if the edge intersect with CMBR (from boolPIndex)
  // prefix sum filter: check if the current edge has any intersection count
  if(psQ1[id+1]!=psQ1[id])
  {
    point P1, P2, Q1, Q2;

    P1.x = polyQX[id];
    P1.y = polyQY[id];
    P2.x = polyQX[(id+1)%sizeQ];
    P2.y = polyQY[(id+1)%sizeQ];

    for(int qid=0; qid<sizeP; qid++){        
      // prefix sum filter: check if the current edge has any intersection count      
      if(psP1[qid+1]!=psP1[qid])
      {
        Q1.x = polyPX[qid];
        Q1.y = polyPY[qid];
        Q2.x = polyPX[(qid+1)%sizeP];
        Q2.y = polyPY[(qid+1)%sizeP];

        // if(gpuLSMF(P1, P2, Q1, Q2))
        {
          // determine intersection or overlap type
          int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
          if(i!=0){
            count1++;
            if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7))){
              nonDegenCount++;
              count2=nonDegenCount;
            }
            else if((id<sizeP && (i==2 || i==4 || i==6 || i==8)) || (id>=sizeP && (i==2 || i==4 || i==6 || i==8)))
              count2=0;

            neighborMapQ[psQ2[id]+count2]=qid;      
          }
        }
      }
    }
  } 
}

/*
-----------------------------------------------------------------
Function to calculate all intersections save them in the correct 
location using prefixsum arrays and make neighbor connections
Returns 
  *intersection arrays with orginal vertices in them x2 (P and Q)
  *neighbor arrays x2 (P and q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCalculateIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX, double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psP2, int *psQ1, int *psQ2, 
                  double *intersectionsP, double *intersectionsQ, double *intersectionsP2, double *intersectionsQ2,
                  int *alphaValuesP, int *alphaValuesQ, int *tmpBucketP, int *alphaSortedIndiciesP,
                  int *neighborP, int *neighborQ, int *neighborP2, int *neighborQ2,
                  int *neighborMapQ /*, int *boolPIndex, int *boolQIndex*/){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, nonDegenCount=0, start, end, localI, neighborQId;

  if(id>=sizeP) return;

  point P1, P2, Q1, Q2;
  int pid=id;

  intersectionsP[psP2[pid]*2]=polyPX[pid];       //consider edge for the intersection array
  intersectionsP[psP2[pid]*2+1]=polyPY[pid];
  intersectionsP2[psP2[pid]*2]=polyPX[pid];       //consider edge for the intersection array
  intersectionsP2[psP2[pid]*2+1]=polyPY[pid];
  alphaValuesP[psP2[pid]]=-100;

  if(id<sizeQ){
    intersectionsQ[psQ2[pid]*2]=polyQX[pid];       //consider edge for the intersection array
    intersectionsQ[psQ2[pid]*2+1]=polyQY[pid];
    intersectionsQ2[psQ2[pid]*2]=polyQX[pid];       //consider edge for the intersection array
    intersectionsQ2[psQ2[pid]*2+1]=polyQY[pid];
  }

  // prefix sum filter: check if the current edge has any intersection count      
  if(psP1[id+1]!=psP1[id])
  // CMBR filter followed by prefix sum filter
  // if(boolPIndex[id] && psP1[id+1]!=psP1[id])
  {
    P1.x = polyPX[pid];
    P1.y = polyPY[pid];
    P2.x = polyPX[(pid+1)%sizeP];
    P2.y = polyPY[(pid+1)%sizeP];

    for(int qid=0; qid<sizeQ; qid++){
      // prefix sum filter: check if the current edge has any intersection count      
      if(psQ1[qid+1]!=psQ1[qid])
      // CMBR filter followed by prefix sum filter
      // if(boolQIndex[qid] && psQ1[qid+1]!=psQ1[qid])
      {
        Q1.x = polyQX[qid];
        Q1.y = polyQY[qid];
        Q2.x = polyQX[(qid+1)%sizeQ];
        Q2.y = polyQY[(qid+1)%sizeQ];

        // if(gpuLSMF(P1, P2, Q1, Q2))
        {
          // determine intersection or overlap type
          int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
          if(i){
            count1++;
            if(i==1 || i==3 || i==5 || i==7){
              nonDegenCount++;
              count2=nonDegenCount;
            }
            else if(i==2 || i==4 || i==6 || i==8)
              count2=0;
            start=psQ2[qid];
            end=psQ2[qid+1];

            if(i!=5){
              // local search to find the index of qid
              for(localI=start; localI<end; ++localI){
                if(pid==neighborMapQ[localI]){
                  neighborQId=localI;
                  neighborP[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborP2[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ2[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  localI=end+2; // break; 
                }
              }
            }else{
              neighborQId=start;
              neighborP[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
              neighborP2[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
              neighborQ[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
              neighborQ2[neighborQId]=psP2[pid]+count2+1;
              
              for(localI=start; localI<end; ++localI){
                if(pid==neighborMapQ[localI]){
                  neighborQId=localI;
                  neighborP[psP2[pid]]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborP2[psP2[pid]]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ[neighborQId]=psP2[pid]+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ2[neighborQId]=psP2[pid]+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  localI=end+2; // break; 
                }
              }
            }
            switch(i) {
              // case X_INTERSECTION:
              // I and I
              case 1:
                I = add(mulScalar((1.0-alpha), P1), mulScalar(alpha, P2));
                intersectionsP[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
                intersectionsP[(psP2[pid]+count2)*2+1]=I.y;
                intersectionsP2[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
                intersectionsP2[(psP2[pid]+count2)*2+1]=I.y;
                alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                intersectionsQ[neighborQId*2]=I.x;       //consider edge for the intersection array
                intersectionsQ[neighborQId*2+1]=I.y;
                intersectionsQ2[neighborQId*2]=I.x;       //consider edge for the intersection array
                intersectionsQ2[neighborQId*2+1]=I.y;
                alphaValuesQ[neighborQId]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
              // X-overlap
              // P1 and I(=P1 I is in Q)
              // I(=Q1 I is in P) and Q1
              case 5:
                intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
                intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
                alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                intersectionsQ[neighborQId*2]=P1.x;    
                intersectionsQ[neighborQId*2+1]=P1.y;
                intersectionsQ2[neighborQId*2]=P1.x;    
                intersectionsQ2[neighborQId*2+1]=P1.y;
                alphaValuesQ[neighborQId]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
              // case T_INTERSECTION_Q:
              // case T_OVERLAP_Q:
              // P1 and I(=P1 is in Q)
              case 2:
              case 6:
                alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                intersectionsQ[neighborQId*2]=P1.x;
                intersectionsQ[neighborQId*2+1]=P1.y;
                intersectionsQ2[neighborQId*2]=P1.x;
                intersectionsQ2[neighborQId*2+1]=P1.y;
                alphaValuesQ[neighborQId]=(int)pow(10, EPSILON_POSITIONS)*beta;
              break;
              // case T_INTERSECTION_P:
              // case T_OVERLAP_P:
              // I(=Q1 is in P) and Q1
              case 3:
              case 7:
                intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
                intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
                alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                alphaValuesQ[psQ2[qid]]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
              // case V_INTERSECTION:
              // case V_OVERLAP:
              // P1 and Q1
              case 4:
              case 8:
                alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
                
                alphaValuesQ[psQ2[qid]]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
            } 
          } 
        }
      }
    }
  
    // --------------------------------------------------------------------------------------------
    // local sort for each edge, start to end
    // --------------------------------------------------------------------------------------------
    start=psP2[pid];
    end=psP2[pid+1];
    // sort intersection vertices in this edge locally
    if((end-start)>2){
      gpuRadixsort(alphaValuesP, tmpBucketP, alphaSortedIndiciesP, start+1, end);
      // using sorted index array, change intersection locations in the array and neighbors
      // decending order JUST FOR TESING
      // for(int i=start+1, j=end-1; i<end; ++i, j--){
      // acending order of alpha values 
      for(int i=start+1, j=start+1; i<end; i++, j++){
        alphaValuesP[i]=tmpBucketP[j];
        intersectionsP[i*2]=intersectionsP2[alphaSortedIndiciesP[j]*2];
        intersectionsP[i*2+1]=intersectionsP2[alphaSortedIndiciesP[j]*2+1];
        neighborP[i]=neighborP2[alphaSortedIndiciesP[j]];
        neighborQ[neighborP2[alphaSortedIndiciesP[j]]-1]=i+1; //+1 is the padding. When reading do -1
        neighborQ2[neighborP2[alphaSortedIndiciesP[j]]-1]=i+1; //updates neighborQ2 as the new original to be used with sorted Q array
      } 
    } 
  // --------------------------------------------------------------------------------------------
  }
}

/*
-----------------------------------------------------------------
Function to save vertices of Q in edge wise sorted order
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuSortPolyQ(
                  int sizeQ, 
                  int *psQ2, 
                  double *intersectionsQ, double *intersectionsQ2,
                  int *alphaValuesQ, int *tmpBucketQ,  int *alphaSortedIndiciesQ,
                  int *neighborP, int *neighborQ, int *neighborQ2){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;

  if(id<sizeQ){
    int start=psQ2[id], end=psQ2[id+1];
    // sort intersection vertices in this edge locally
    if((end-start)>2){
      gpuRadixsort(alphaValuesQ, tmpBucketQ, alphaSortedIndiciesQ, start+1, end);
      // using sorted index array, change intersection locations in the array and neighbors
      // decending order JUST FOR TESING
      // for(int i=start+1, j=end-1; i<end; ++i, j--){
      // acending order of alpha values 
      for(int i=start+1, j=start+1; i<end; i++, j++){
        alphaValuesQ[i]=tmpBucketQ[j];////////////////?????????????????????? need to swap alpha too!!!
        // (x,y,alpha) tuple change in sorted order
        intersectionsQ[i*2]=intersectionsQ2[alphaSortedIndiciesQ[j]*2];
        intersectionsQ[i*2+1]=intersectionsQ2[alphaSortedIndiciesQ[j]*2+1];
        //neighbor array update
        neighborQ[i]=neighborQ2[alphaSortedIndiciesQ[j]];
        neighborP[neighborQ2[alphaSortedIndiciesQ[j]]-1]=i+1; //+1 is the padding. When reading do -1 //[]= i+1
      } 
    } 
  }
}

/*
-----------------------------------------------------------------
Function to calculate initial label
Returns 
  *initial labels x2 (P and Q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCalculateInitLabel(
                int sizeP, int *psP2,
                double *intersectionsP, double *intersectionsQ, int *alphaValuesP, 
                int *neighborP,
                int sizeNP, int sizeNQ, int *initLabelsP, int *initLabelsQ){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  int pid=id;
  if(id>=sizeP) return;
  int start=psP2[pid], end=psP2[pid+1];
  // int start=psP2[id], end=psP2[id+1];
  int tmpId, nId, pMNId, pPNId;
  point pM, pP, qM, qP, current;
  int qMType, qPType, tmpIniLabel;
  int i;
  for(i=start; i<end; i++){
    initLabelsP[i]=-100;
    if(alphaValuesP[i]!=-100){    //consider intersections only
      current.x=intersectionsP[i*2]; 
      current.y=intersectionsP[i*2+1]; 
      tmpId=getCircularId(i-1, sizeNP);
      // determine local configuration at this intersection vertex
      pM.x=intersectionsP[tmpId*2];                // P-, predecessor of I on P
      pM.y=intersectionsP[tmpId*2+1];                // P-, predecessor of I on P
      // if(intersectionsP[tmpId*2+2]!=-100)
      if(alphaValuesP[tmpId]!=-100)
        pMNId=neighborP[tmpId]-1; //get neighbor id of P_m vertex
      else pMNId=-100;

      tmpId=getCircularId(i+1, sizeNP);
      pP.x=intersectionsP[tmpId*2];                // P+, successor of I on P
      pP.y=intersectionsP[tmpId*2+1];                // P+, successor of I on P
      if(alphaValuesP[tmpId]!=-100)
        pPNId=neighborP[tmpId]-1; //get neighbor id of P_p vertex
      else pPNId=-100;

      // nId=getNeighborIndex(i, neighborMapP, neighborQ);
      nId=neighborP[i]-1;
      tmpId=getCircularId(nId-1, sizeNQ);
      qM.x=intersectionsQ[tmpId*2];     // Q-, predecessor of I on Q
      qM.y=intersectionsQ[tmpId*2+1];     // Q-, predecessor of I on Q
      qMType=oracle(pMNId, pPNId, tmpId, qM, pM, current, pP);

      tmpId=getCircularId(nId+1, sizeNQ);
      qP.x=intersectionsQ[tmpId*2];     // Q+, successor of I on P
      qP.y=intersectionsQ[tmpId*2+1];     // Q+, successor of I on P
      qPType=oracle(pMNId, pPNId, tmpId, qP, pM, current, pP);

      tmpIniLabel=getInitialLabel(qMType, qPType);
      initLabelsP[i]=tmpIniLabel;
      initLabelsQ[nId]=tmpIniLabel;
    }
  }
}

/*
-----------------------------------------------------------------
Function to count how many intersection points and prefix sums
Returns 
  *count of non degenerate vertices x2 (P and Q)
  *intersection points with non degenrate vertices included x2
  *neighbor map x2
  *neighbor arrays x2
  *initial labels x2
Neighbor of a vertex (assume index i) in P can be read in O(1) time using
  neighborQ[neighborMapP[i]]
  for Q
    neighborP[neighborMapQ[i]]
Runs in CPU
Called from Host
-------------------------------------------------------------------
*/
void calculateIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, double *cmbr,
                  int *countNonDegenIntP, int *countNonDegenIntQ, 
                  double **intersectionsP, double **intersectionsQ, int **alphaValuesP, int **alphaValuesQ,
                  int **initLabelsP, int **initLabelsQ,
                  int **neighborP, int **neighborQ){
    double *dev_polyPX, *dev_polyPY, *dev_polyQX, *dev_polyQY;
    int *dev_psP1, *dev_psP2, *dev_psQ1, *dev_psQ2, *dev_boolPsPX, *dev_boolPsQX, *dev_boolPX, *dev_boolQX;
    int psP1[sizeP+1], psP2[sizeP+1], psQ1[sizeQ+1], psQ2[sizeQ+1];
    int boolPsPX[sizeP+1], boolPsQX[sizeQ+1];
    hipEvent_t kernelStart0, kernelStart1, kernelStart12, kernelStart2, kernelStart3, kernelStart4, kernelStart5, kernelStart6, kernelStart7, kernelStart8;
    hipEvent_t kernelStop0, kernelStop1, kernelStop12, kernelStop2, kernelStop3, kernelStop4, kernelStop5, kernelStop6, kernelStop7, kernelStop8;
    int countCMBRP,countCMBRQ, sum;

    // printf("cmbr %f %f %f %f\n",*(cmbr+0), *(cmbr+1), *(cmbr+2), *(cmbr+3));
    
    // Phase1: Count intersections in each block. Create prefix sums to find local locations in each thread 
    // Allocate memory in device 
    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart0);
        hipEventCreate(&kernelStop0);
    }
    hipMalloc((void **) &dev_polyPX, sizeP*sizeof(double));
    hipMalloc((void **) &dev_polyPY, sizeP*sizeof(double));
    hipMalloc((void **) &dev_polyQX, sizeQ*sizeof(double));
    hipMalloc((void **) &dev_polyQY, sizeQ*sizeof(double));
    hipMalloc((void **) &dev_psP1, (sizeP+1)*sizeof(int));
    hipMalloc((void **) &dev_psP2, (sizeP+1)*sizeof(int));
    hipMalloc((void **) &dev_psQ1, (sizeQ+1)*sizeof(int));
    hipMalloc((void **) &dev_psQ2, (sizeQ+1)*sizeof(int));

    // hipMalloc((void **) &dev_boolPX, sizeP*sizeof(int));
    // hipMalloc((void **) &dev_boolQX, sizeQ*sizeof(int));
    hipMalloc((void **) &dev_boolPsPX, (sizeP+1)*sizeof(int));
    hipMalloc((void **) &dev_boolPsQX, (sizeQ+1)*sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_polyPX, polyPX, sizeP*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_polyPY, polyPY, sizeP*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_polyQX, polyQX, sizeQ*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_polyQY, polyQY, sizeQ*sizeof(double), hipMemcpyHostToDevice);

    int blocksPerGrid=((sizeP+sizeQ) + xThreadPerBlock - 1) / xThreadPerBlock;
    int xBlocksPerGrid=(blocksPerGrid + yBlockPerGrid - 1) / yBlockPerGrid;

    int blocksPerGridQ=(sizeQ + xThreadPerBlock - 1) / xThreadPerBlock;
    int xBlocksPerGridQ=(blocksPerGridQ + yBlockPerGrid - 1) / yBlockPerGrid;
    int blocksPerGridP=(sizeP + xThreadPerBlock - 1) / xThreadPerBlock;
    int xBlocksPerGridP=(blocksPerGridP + yBlockPerGrid - 1) / yBlockPerGrid;
    
    // ******size_t number_of_blocks = N/threads_per_block + (size_t)(N % threads_per_block != 0);
    dim3 dimBlock(xThreadPerBlock, yThreadPerBlock, 1);
    dim3 dimGridP(xBlocksPerGridP, yBlockPerGrid, 1); 
    dim3 dimGridQ(xBlocksPerGridQ, yBlockPerGrid, 1); 


    // CMBR filter 

    if(DEBUG_TIMING) hipEventRecord(kernelStart0);
    gpuCMBRFilter<<<dimGridP, dimBlock>>>(
                dev_polyPX, dev_polyPY, 
                cmbr[0], cmbr[1], cmbr[2], cmbr[3],
                sizeP, dev_boolPsPX, dev_psP1, dev_psP2);
    gpuCMBRFilter<<<dimGridQ, dimBlock>>>(
                dev_polyQX, dev_polyQY, 
                cmbr[0], cmbr[1], cmbr[2], cmbr[3],
                sizeQ, dev_boolPsQX, dev_psQ1, dev_psQ2);

    if(DEBUG_TIMING) hipEventRecord(kernelStop0);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop0);

    hipDeviceSynchronize();
  
    if(DEBUG_INFO_PRINT){
      hipMemcpy(&boolPsPX, dev_boolPsPX, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&boolPsQX, dev_boolPsQX, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);
      // count how many edges overlap with CMBRs
      countCMBRP=0;
      for(int x=0; x<sizeP; ++x) if(boolPsPX[x]) countCMBRP++;
      printf("\nP overlap count with CMBR %d ",countCMBRP);
      countCMBRQ=0;
      for(int x=0; x<sizeQ; ++x) if(boolPsQX[x]) countCMBRQ++;
      printf("Q overlap count with CMBR %d \n\n",countCMBRQ);
    }

    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart1);
        hipEventCreate(&kernelStop1);
    }

    if(DEBUG_TIMING) hipEventRecord(kernelStart1);
    gpuCountIntersections<<<dimGridQ, dimBlock>>>(
          dev_polyQX, dev_polyQY, 
          dev_polyPX, dev_polyPY, 
          sizeQ, sizeP,
          dev_psQ1, dev_psQ2, dev_boolPsQX);
    
    if(DEBUG_TIMING) hipEventRecord(kernelStop1);
    if(DEBUG_TIMING) hipEventSynchronize(kernelStop1);


    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart12);
        hipEventCreate(&kernelStop12);
    }
    if(DEBUG_TIMING) hipEventRecord(kernelStart12);

    gpuCountIntersections<<<dimGridP, dimBlock>>>(
          dev_polyPX, dev_polyPY, 
          dev_polyQX, dev_polyQY, 
          sizeP, sizeQ,
          dev_psP1, dev_psP2, dev_boolPsPX);

    if(DEBUG_TIMING) hipEventRecord(kernelStop12);

    hipDeviceSynchronize();

    hipFree(dev_boolPsPX);
    hipFree(dev_boolPsQX);

    dim3 dimGrid2(xBlocksPerGrid, yBlockPerGrid, 1);

    hipMemcpy(&psP1, dev_psP1, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psP2, dev_psP2, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psQ1, dev_psQ1, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psQ2, dev_psQ2, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop12);
    hipDeviceSynchronize();

    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart2);
        hipEventCreate(&kernelStop2);
    }
    if(DEBUG_TIMING) hipEventRecord(kernelStart2);
    thrust::exclusive_scan(thrust::host, psP1, psP1 + sizeP+1, psP1);   //sizeP location contains the total size of the count1
    thrust::exclusive_scan(thrust::host, psP2, psP2 + sizeP+1, psP2);
    thrust::exclusive_scan(thrust::host, psQ1, psQ1 + sizeQ+1, psQ1);   //sizeQ location contains the total size of the count1
    thrust::exclusive_scan(thrust::host, psQ2, psQ2 + sizeQ+1, psQ2);
    if(DEBUG_TIMING) hipEventRecord(kernelStop2);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop2);

    hipDeviceSynchronize();

    //Phase2: NEW- Fill neighborMap
    int *dev_neighborMapQ;
    int *neighborMapQ;
    *countNonDegenIntP=psP2[sizeP];
    *countNonDegenIntQ=psQ2[sizeQ];

    if(DEBUG_INFO_PRINT){
      printf("Non-degen count P %d *****--- Q %d\n", *countNonDegenIntP-sizeP, *countNonDegenIntQ-sizeQ);
      printf("Intersection count P %d *****--- Q %d\n", psP1[sizeP], psQ1[sizeQ]);
    }

    dim3 dimGrid(xBlocksPerGrid, yBlockPerGrid, 1);

    neighborMapQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));

    hipMalloc((void **) &dev_neighborMapQ, *countNonDegenIntQ*sizeof(int));

    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart3);
        hipEventCreate(&kernelStop3);
    }
    hipMemcpy(dev_psP1, psP1, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_psP2, psP2, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_psQ1, psQ1, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_psQ2, psQ2, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);

    if(DEBUG_TIMING) hipEventRecord(kernelStart3);

    gpuNeighborMap<<<dimGridQ, dimBlock>>>(
            dev_polyPX, dev_polyPY, 
            dev_polyQX, dev_polyQY, 
            sizeP, sizeQ,  
            dev_psP1, dev_psQ1, dev_psQ2,
            dev_neighborMapQ);
    if(DEBUG_TIMING) hipEventRecord(kernelStop3);
  
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop3);

  // Phase 3: Calcualte intersections and save them in the arrays. Make neighbor connections
  int countIntersections=psP1[sizeP];

  int *alphaSortedIndiciesP, *alphaSortedIndiciesQ;
  double *dev_intersectionsP, *dev_intersectionsQ, *dev_intersectionsP2, *dev_intersectionsQ2;
  int *dev_neighborP, *dev_neighborQ, *dev_neighborP2, *dev_neighborQ2;
  int *dev_initLabelsP, *dev_initLabelsQ;
  int *dev_alphaValuesP, *dev_alphaValuesQ, *dev_tmpBucketP, *dev_tmpBucketQ, *dev_alphaSortedIndiciesP, *dev_alphaSortedIndiciesQ;

  *intersectionsP=(double *)malloc(*countNonDegenIntP*2*sizeof(double));
  *intersectionsQ=(double *)malloc(*countNonDegenIntQ*2*sizeof(double));
  *alphaValuesP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *alphaValuesQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  alphaSortedIndiciesP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  alphaSortedIndiciesQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  *initLabelsP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *initLabelsQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  *neighborP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *neighborQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));

  for(int i=0; i<*countNonDegenIntQ; ++i){
    *(*initLabelsQ+i)=-100;
    *(*alphaValuesQ+i)=-100;
  }

  hipDeviceSynchronize();

  // Allocate memory in device 
  hipMalloc((void **) &dev_intersectionsP, *countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsP2, *countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ, *countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ2, *countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_alphaValuesP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaValuesQ, *countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_tmpBucketP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_tmpBucketQ, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesQ, *countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_neighborP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborP2, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborQ, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_neighborQ2, *countNonDegenIntQ*sizeof(int));

  hipMemcpy(dev_alphaValuesQ, *alphaValuesQ, *countNonDegenIntQ*sizeof(int), hipMemcpyHostToDevice);

  
  if(DEBUG_TIMING){
    hipEventCreate(&kernelStart4);
    hipEventCreate(&kernelStop4);
  }

  if(DEBUG_TIMING) hipEventRecord(kernelStart4);
  gpuCalculateIntersections<<<dimGridP, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ, 
        dev_psP1, dev_psP2, dev_psQ1, dev_psQ2, 
        dev_intersectionsP, dev_intersectionsQ, dev_intersectionsP2, dev_intersectionsQ2,
        dev_alphaValuesP, dev_alphaValuesQ, dev_tmpBucketP, dev_alphaSortedIndiciesP,
        dev_neighborP, dev_neighborQ, dev_neighborP2, dev_neighborQ2,
        dev_neighborMapQ);
  if(DEBUG_TIMING) hipEventRecord(kernelStop4);
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop4);

  hipDeviceSynchronize();

  hipFree(dev_polyPX);
  hipFree(dev_polyPY);
  hipFree(dev_polyQX);
  hipFree(dev_polyQY);
  hipFree(dev_neighborMapQ);
  hipFree(dev_intersectionsP2);
  hipFree(dev_tmpBucketP);
  hipFree(dev_alphaSortedIndiciesP);
  hipFree(dev_neighborP2);
  hipFree(dev_psP1);
  hipFree(dev_psQ1);

  if(DEBUG_TIMING){
    hipEventCreate(&kernelStart5);
    hipEventCreate(&kernelStop5);
  }
  if(DEBUG_TIMING) hipEventRecord(kernelStart5);
  gpuSortPolyQ<<<dimGridQ, dimBlock>>>(
        sizeQ, 
        dev_psQ2, 
        dev_intersectionsQ, dev_intersectionsQ2,
        dev_alphaValuesQ, dev_tmpBucketQ,  dev_alphaSortedIndiciesQ,
        dev_neighborP, dev_neighborQ, dev_neighborQ2);
  if(DEBUG_TIMING) hipEventRecord(kernelStop5);
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop5);

  hipDeviceSynchronize();

  hipFree(dev_psQ2);
  hipFree(dev_intersectionsQ2);
  hipFree(dev_tmpBucketQ);
  hipFree(dev_alphaSortedIndiciesQ);
  hipFree(dev_neighborQ2);

  // Phase4: Inital label classificaiton
  hipMalloc((void **) &dev_initLabelsP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_initLabelsQ, *countNonDegenIntQ*sizeof(int));

  hipMemcpy(dev_initLabelsQ, *initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyHostToDevice);
 
  // negative alpha values are not handled explicitly since they are original vertices
  // ******No need to copy alpha values since they are only used to sort edge wise******
  // hipMemcpy(alphaSortedIndicies, dev_alphaSortedIndicies, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  if(DEBUG_TIMING){
    hipEventCreate(&kernelStart6);
    hipEventCreate(&kernelStop6);
  }

  if(DEBUG_TIMING) hipEventRecord(kernelStart6);
  gpuCalculateInitLabel<<<dimGridP, dimBlock>>>(
      sizeP,  dev_psP2,
      dev_intersectionsP, dev_intersectionsQ, dev_alphaValuesP,
      dev_neighborP,
      *countNonDegenIntP, *countNonDegenIntQ, dev_initLabelsP, dev_initLabelsQ);
  if(DEBUG_TIMING) hipEventRecord(kernelStop6);

  hipMemcpy(*intersectionsP, dev_intersectionsP, *countNonDegenIntP*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*intersectionsQ, dev_intersectionsQ, *countNonDegenIntQ*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborP, dev_neighborP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborQ, dev_neighborQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
 
  hipMemcpy(*initLabelsP, dev_initLabelsP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*initLabelsQ, dev_initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesP, dev_alphaValuesP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesQ, dev_alphaValuesQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop6);
  
  hipDeviceSynchronize();

  float kernelTiming0=0, kernelTiming1=0, kernelTiming12=0, kernelTiming2=0, kernelTiming3=0, kernelTiming4=0, kernelTiming5=0, kernelTiming6=0;
  if(DEBUG_TIMING){
    hipEventElapsedTime(&kernelTiming0, kernelStart0, kernelStop0);
    hipEventElapsedTime(&kernelTiming1, kernelStart1, kernelStop1);
    hipEventElapsedTime(&kernelTiming12, kernelStart12, kernelStop12);
    hipEventElapsedTime(&kernelTiming2, kernelStart2, kernelStop2);
    hipEventElapsedTime(&kernelTiming3, kernelStart3, kernelStop3);
    hipEventElapsedTime(&kernelTiming4, kernelStart4, kernelStop4);
    hipEventElapsedTime(&kernelTiming5, kernelStart5, kernelStop5);
    hipEventElapsedTime(&kernelTiming6, kernelStart6, kernelStop6);
    // printf("gpuCMBR kernel exe time(microsecond) %f\n", kernelTiming0*1000);
    // printf("gpuCountIntersections kernel exe time(microsecond) %f\n", kernelTiming1*1000);
    // printf("gpuCountIntersections2 kernel exe time(microsecond) %f\n", kernelTiming12*1000);
    // printf("prefixsum kernels exe time(microsecond) %f\n", kernelTiming2*1000);
    // printf("gpuNeighborMap kernel exe time(microsecond) %f\n", kernelTiming3*1000);
    // printf("gpuCalculateIntersections kernel exe time(microsecond) %f\n", kernelTiming4*1000);
    // printf("gpuSortPolyQ kernel exe time(microsecond) %f\n", kernelTiming5*1000);
    // printf("gpuCalculateInitLabel kernel exe time(microsecond) %f\n\n", kernelTiming6*1000);
    
    printf("%f, %f, %f, %f, %f, %f, ", (kernelTiming1*1000 + kernelTiming12*1000), 
          kernelTiming2*1000, kernelTiming3*1000, kernelTiming4*1000, 
          kernelTiming5*1000, kernelTiming6*1000);
  }

  int limitP=10;
  int limitQ=10;

  hipFree(dev_psP2);
  hipFree(dev_intersectionsP);
  hipFree(dev_intersectionsQ);
  hipFree(dev_alphaValuesP);
  hipFree(dev_alphaValuesQ);
  hipFree(dev_neighborP);
  hipFree(dev_neighborQ);
  hipFree(countNonDegenIntP);
  hipFree(countNonDegenIntQ);
  hipFree(dev_initLabelsP);
  hipFree(dev_initLabelsQ);

  // hipFree(dev_polyPX);
  // hipFree(dev_polyPY);
  // hipFree(dev_polyQX);
  // hipFree(dev_polyQY);
}