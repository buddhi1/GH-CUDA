#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <hip/hip_cooperative_groups.h>

#include "lib/constants.h"

typedef struct{
  double x, y;
} point;

__device__ double A(const point& P, const point& Q, const point& R){
	return (Q.x-P.x) * (R.y-P.y) - (Q.y-P.y) * (R.x-P.x);
}

// difference of two 2D points
__device__ point sub(const point& a, const point& b){ 
  point r;
  r.x=a.x-b.x;
  r.y=a.y-b.y;
  return r; 
}

// add two 2D points
__device__ point add(const point& a, const point& b){ 
  point r;
  r.x=a.x+b.x;
  r.y=a.y+b.y;
  return r; 
}

// multiply two 2D points
__device__ double mul(const point& a, const point& b){ 
  point r;
  r.x=a.x*b.x;
  r.y=a.y*b.y;
  return (r.x+r.y); 
}

// multiply scalar with 2D points
__device__ point mulScalar(const double c, const point& b){ 
  point r;
  r.x=c*b.x;
  r.y=c*b.y;
  return r; 
}

// find min
__device__ double getMin(double a, double b){
  if(a<b) return a;
  return b;
}

// find max
__device__ double getMax(double a, double b){
  if(a<b) return b;
  return a;
}

/*
-----------------------------------------------------------------
Function to returns the start index of the current id's 
intersections
Returns the intersection starting index
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int getIntersectionStartIndex(int id, int *ps1){
  if(id==0) return 0;
  else return ps1[id];
}

/*
-----------------------------------------------------------------
Function to return intersection  type
Returns the type of the intersection
Runs in GPU
Called from Device
  NO_INTERSECTION, //0
  X_INTERSECTION,  //1
  T_INTERSECTION_Q, //2
  T_INTERSECTION_P, //3
  V_INTERSECTION, //4
  X_OVERLAP,      //5
  T_OVERLAP_Q,    //6
  T_OVERLAP_P,    //7
  V_OVERLAP       //8
-------------------------------------------------------------------
*/
__device__ int getIntersectType(
            const point& P1, const point& P2, 
            const point& Q1, const point& Q2,  
            double& alpha, double& beta){
	double AP1 = A(P1,Q1,Q2);
	double AP2 = A(P2,Q1,Q2);

	if (fabs(AP1-AP2) > EPSILON){
		// from here: [P1,P2] and [Q1,Q2] are not parallel
		// analyse potential intersection
		double AQ1 = A(Q1,P1,P2);
		double AQ2 = A(Q2,P1,P2);
		// compute alpha and beta
		alpha = AP1 / (AP1-AP2);
		beta  = AQ1 / (AQ1-AQ2);
		// classify alpha
		bool alpha_is_0 = false;
		bool alpha_in_0_1 = false;
		if ( (alpha > EPSILON) && (alpha < 1.0-EPSILON) )
			alpha_in_0_1 = true;
		else
			if (fabs(alpha) <= EPSILON)
				alpha_is_0 = true;
		// classify beta
		bool beta_is_0 = false;
		bool beta_in_0_1 = false;
		if ( (beta > EPSILON) && (beta < 1.0-EPSILON) )
			beta_in_0_1 = true;
		else
			if (fabs(beta) <= EPSILON)
				beta_is_0 = true;
		// distinguish intersection types
		if (alpha_in_0_1 && beta_in_0_1) return (1);  // return (X_INTERSECTION);
		if (alpha_is_0 && beta_in_0_1) return (2);    // return (T_INTERSECTION_Q);
		if (beta_is_0 && alpha_in_0_1) return (3);    // return (T_INTERSECTION_P);
		if (alpha_is_0 && beta_is_0) return (4);      // return (V_INTERSECTION);
	}else if (fabs(AP1) < EPSILON){
			// from here: [P1,P2] and [Q1,Q2] are collinear
			// analyse potential overlap			
      point dP = sub(P2, P1);
			point dQ = sub(Q2, Q1);
			point PQ = sub(Q1, P1);
			alpha = mul(PQ,dP) / mul(dP,dP);
			beta = -mul(PQ,dQ) / mul(dQ,dQ);
			// classify alpha
			bool alpha_is_0 = false;
			bool alpha_in_0_1 = false;
			bool alpha_not_in_0_1 = false;
			if ((alpha > EPSILON) && (alpha < 1.0-EPSILON))
				alpha_in_0_1 = true;
			else
				if (fabs(alpha) <= EPSILON)
					alpha_is_0 = true;
				else
					alpha_not_in_0_1 = true;
			// classify beta
			bool beta_is_0 = false;
			bool beta_in_0_1 = false;
			bool beta_not_in_0_1 = false;
			if ((beta > EPSILON) && (beta < 1.0-EPSILON))
				beta_in_0_1 = true;
			else
				if (fabs(alpha) <= EPSILON)
					beta_is_0 = true;
				else
					beta_not_in_0_1 = true;

			// distinguish intersection types
			if (alpha_in_0_1 && beta_in_0_1) return (5);      // return (X_OVERLAP);
			if (alpha_not_in_0_1 && beta_in_0_1) return (6);  // return (T_OVERLAP_Q);
			if (beta_not_in_0_1 && alpha_in_0_1) return (7);  // return (T_OVERLAP_P);
			if (alpha_is_0 && beta_is_0) return (8);          // return (V_OVERLAP);
		}
  return (0);	// return (NO_INTERSECTION); 
}

/*
-----------------------------------------------------------------
Function to get circular id of a given id 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int getCircularId(int id, int maxCount){
  if(maxCount==id) return 0;
  else if(id==-1) return maxCount-1;
  else return id;
}

/*
-----------------------------------------------------------------
Function to get relative position type
Runs in GPU
Called from Device
0 -> LEFT,
1 -> RIGHT,
2 -> IS_P_m,
3 -> IS_P_p
-------------------------------------------------------------------
*/
__device__ int oracle(int pMNId, int pPNId, int qId, const point& Q, const point& P1, const point& P2, const point& P3) {
  // is Q linked to P1 ?
  if(pMNId!=-100 && pMNId==qId) return 2;
  // is Q linked to P2 ?
  else if(pPNId!=-100 && pPNId==qId) return 3;
  // check relative position of Q with respect to chain (P1,P2,P3)
  double s1 = A(Q, P1, P2);
  double s2 = A(Q, P2, P3);
  double s3 = A(P1, P2, P3);
  if(s3>0){ 
    // chain makes a left turn
    if (s1>0 && s2>0)
      return 0;
    else
      return 1;
  }else{
    // chain makes a right turn (or is straight)
    if(s1<0 && s2<0)
      return 1;
    else
      return 0;
  }
}

/*
-----------------------------------------------------------------
Function to get initial classification label
Runs in GPU
Called from Device
Intersection Labels
0  NONE,
1  CROSSING,
2  BOUNCING,
3  LEFT_ON,
4  RIGHT_ON,
5  ON_ON,
6  ON_LEFT,
7  ON_RIGHT,
8  DELAYED_CROSSING,
9  DELAYED_BOUNCING
-------------------------------------------------------------------
*/
__device__ int getInitialLabel(int qMType, int qPType){
  // check non-overlapping cases
  if((qMType==0  && qPType==1)||(qMType==1 && qPType==0)){
    return 1;
  }
  if((qMType==0  && qPType==0)||(qMType==1 && qPType==1)){
    return 2;
  }
  // check overlapping cases
  if(((qPType==3) && (qMType==1))||((qMType==3) && (qPType==1))) return 3;
  if(((qPType==3) && (qMType==0))||((qMType==3) && (qPType==0))) return 4;
  if(((qPType==3) && (qMType==2))||((qMType==3) && (qPType==2))) return 5;
  if(((qMType==2) && (qPType==1))||((qPType==2) && (qMType==1))) return 6;
  if(((qMType==2) && (qPType==0))||((qPType==2) && (qMType==0))) return 7;
  else return -102;
}

/*
-----------------------------------------------------------------
Function to get a given double value within tolerance 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/ //**************************** WRONG FUNCTION. Need to do it in a different way ************
// __device__ double getValueTolarence(double val){
//   if(val<EPSILON)
//     return 0.0;
//   return val;
// }

/*
-----------------------------------------------------------------
Function to do counting sort of arr[] according to
  the digit represented by exp.
Returns sorted by single base digit
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ void gpuCountSort(int arr[], int tmpBucket[], int sortedIndicies[], int start, int end, int exp){
  int *output=tmpBucket; // used to track indices w.r.t original araay values
  int i, count[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  // Store count of occurrences in count[]
  for(i=start; i<end; i++){
    *(output+i)=sortedIndicies[i];
    count[(arr[*(output+i)] / exp) % 10]++;
  }
  // count prefix sum contains actual positions
  for(i=1; i<10; i++){
    count[i] += count[i - 1];
  }
  // Build the output array indices
  for(i=end-1; i>=start; i--){
    sortedIndicies[start+(count[(arr[*(output+i)] / exp) % 10]-1)]=*(output+i);
    count[(arr[*(output+i)] / exp) % 10]--;
  }
}

/*
-----------------------------------------------------------------
Function that sorts arr[] of size n using Radix Sort
Returns sorted array
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ void gpuRadixsort(int arr[], int tmpBucket[], int alphaSortedIndicies[], int start, int end){  
  // Do counting sort for every digit. Note that instead
  // of passing digit number, exp is passed. exp is 10^i
  // where i is current digit number
  int i, exp=1;
  for(i=start; i<end; i++){
      alphaSortedIndicies[i]=i;
  }
  for (i=1; i<=EPSILON_POSITIONS; i++){
    gpuCountSort(arr, tmpBucket, alphaSortedIndicies, start, end, exp);
    exp*=10;
  }
  // record sorted alpha values in tmpBucket
  for(i=start; i<end; ++i)
    tmpBucket[i]=arr[alphaSortedIndicies[i]];
}

/*
-----------------------------------------------------------------
Function to return vertex 2 of a given vertex 1
Returns index of vertex 2 
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int gpuGetVertex2Index(int vertex1Index, int polySize[], int polyId){
  if(vertex1Index<polySize[polyId+1]-1) return vertex1Index+1;
  else if(vertex1Index=polySize[polyId+1]-1) return polySize[polyId];
}

/*
-----------------------------------------------------------------
Function: iterative search 
Returns location of x in given array arr[l..r] if present,
  otherwise -1
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int gpuSearchPolygonId(int arr[], int numPol, int x){
  for(int i=0; i<numPol; ++i){
    if(arr[i]<=x && arr[i+1]>x)
      return i;
  }
  return -1;
}

/*
-----------------------------------------------------------------
Function to check if there is a overlap between given 2 edges 
Returns 1 if there is a overlap; else 0
Runs in GPU
Called from Device
-------------------------------------------------------------------
*/
__device__ int gpuLSMF(point P1, point P2, point Q1, point Q2){
  double minPX=P1.x, minPY=P1.y;
  double maxPX=P2.x, maxPY=P2.y;
  double minQX=Q1.x, minQY=Q1.y;
  double maxQX=Q2.x, maxQY=Q2.y;
  // this toggle way optimizes this computation well compared to using 8 min max calls seperately
  if(minPX>P2.x){
    minPX=P2.x;
    maxPX=P1.x;
  }
  if(minPY>P2.y){
    minPY=P2.y;
    maxPY=P1.y;
  }
  if(minQX>Q2.x){
    minQX=Q2.x;
    maxQX=Q1.x;
  }
  if(minQY>Q2.y){
    minQY=Q2.y;
    maxQY=Q1.y;
  }
  // check intersection between MBRs
  if(minPX>maxQX || maxPX<minQX) return 0;
  if(minPY>maxQY || maxPY<minQY) return 0;
  return 1;
}

/*
-----------------------------------------------------------------
Function to check if edegs are intersecting with the CMBR
Return prefix sum arrays.
  if a marked boolean array if the edges are intersecting with it
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCMBRFilter(
                double *polyX, double *polyY, 
                double cmbrMinX, double cmbrMinY, double cmbrMaxX, double cmbrMaxY,
                int size, int *boolPs, int *ps1, int *ps2){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  
  if(id>size) return;
  
  point P1, P2;
  P1.x=polyX[id];
  P1.y=polyY[id];
  P2.x=polyX[(id+1)%size];
  P2.y=polyY[(id+1)%size];

  double minX=getMin(P1.x, P2.x), minY=getMin(P1.y, P2.y);
  double maxX=getMax(P1.x, P2.x), maxY=getMax(P1.y, P2.y);

  boolPs[id]=1;
  ps1[id]=0;
  ps2[id]=1; //by default paren is in the list. Hence the initial value
  if(minX>cmbrMaxX || maxX<cmbrMinX) boolPs[id]=0;
  if(minY>cmbrMaxY || maxY<cmbrMinY) boolPs[id]=0;
  // if(boolPs[id]!=1) printf("/// %d\n", id);
}

/*
-----------------------------------------------------------------
Function to record all indicies which intersects with CMBR 
Return prefix sum arrays.
  index arrays
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuSaveCMBRIntersectedIndicies(
                double *polyX, double *polyY, 
                double cmbrMinX, double cmbrMinY, double cmbrMaxX, double cmbrMaxY,
                int size, int *boolPol, int *boolPs){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  
  if(id>size) return;
  
  point P1, P2;
  P1.x=polyX[id];
  P1.y=polyY[id];
  P2.x=polyX[(id+1)%size];
  P2.y=polyY[(id+1)%size];

  double minX=getMin(P1.x, P2.x), minY=getMin(P1.y, P2.y);
  double maxX=getMax(P1.x, P2.x), maxY=getMax(P1.y, P2.y);

  int intersect=1;
  if(minX>cmbrMaxX || maxX<cmbrMinX) intersect=0;
  if(minY>cmbrMaxY || maxY<cmbrMinY) intersect=0;
  if(intersect){
    boolPol[boolPs[id]]=id;
    // if(boolPs[id]!=id) printf("Error %d %d \n", id, boolPs[id]);
  }
}

/*
-----------------------------------------------------------------
Function to count all intersections. 
Return prefix sum arrays.
  *prefix sum of count of all intersection vertices x2 (P and Q)
  *prefix sum of count of all intersection vertices excluding 
   degenerate cases x2 (P and Q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*//*
__global__ void gpuCountIntersections1(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psP2, int *psQ1, int *psQ2){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=sizeQ;
  double *poly1X=polyPX, *poly1Y=polyPY, *poly2X=polyQX, *poly2Y=polyQY;
  if(id>=sizeP+sizeQ) return;
  // printf("my id %d tx%d bx%d by%d bdx%d bdy%d gdx%d gdy%d\n", id, threadIdx.x, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
  point P1, P2, Q1, Q2;
  int pid=id;
  if(id>=sizeP){
    size=sizeP;
    poly1X=polyQX; 
    poly1Y=polyQY; 
    poly2X=polyPX;
    poly2Y=polyPY;
    pid=id-sizeP;
  }
  P1.x = poly1X[pid];
  P1.y = poly1Y[pid];
  //polygon1 is P and polygon2 is Q
  if(pid==id && pid==sizeP-1){
    P2.x = poly1X[0];
    P2.y = poly1Y[0];
    // printf("sp %d\n", pid);
  }else if(pid!=id && pid == sizeQ-1){ //polygon2 is P and polygon1 is Q
    P2.x = poly1X[0];
    P2.y = poly1Y[0];
    // printf("sp %d\n", pid);
  } else { //no need reset. Normal case
    P2.x = poly1X[pid+1];
    P2.y = poly1Y[pid+1];
  }

  for(int qid=0; qid<size; qid++){
    Q1.x = poly2X[qid];
    Q1.y = poly2Y[qid];

    // reset P2 vertex of last edge to first vertex
    if(qid == size-1){
      Q2.x = poly2X[0];
      Q2.y = poly2Y[0];
    }else{
      Q2.x = poly2X[qid+1];
      Q2.y = poly2Y[qid+1];
    }
    // determine intersection or overlap type
    int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
    // if (id==0) printf("id %d (%f %f) (%f %f) (%f %f) (%f %f) i %d\n", id, P1.x, P1.y, P2.x, P2.y, Q1.x, Q1.y, Q2.x, Q2.y, i);
    if(i!=0){
      count1++;
      if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7)))
        count2++;
    }
  }
  count2++; //represent the parent vertex 
  if(id<sizeP){
    psP1[pid]=count1;
    psP2[pid]=count2;
    // printf("id %d count1 %d count2 %d\n", id, count1, count2);
  } else{
    psQ1[pid]=count1;
    psQ2[pid]=count2;
    // printf("id %d count1 %d count2 %d\n", id, count1, count2);
  }
}*/
/*
__global__ void gpuCountIntersections2(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, int maxID,
                  int *psP1, int *psP2, int *psQ1, int *psQ2){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  int idx=threadIdx.x;  
  __shared__ double poly2X_shared[MAX_POLY2_SIZE+1], poly2Y_shared[MAX_POLY2_SIZE+1] //+1 for halo next;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=0, size2=sizeQ, checkCount=0;
  double *poly1X=polyPX, *poly1Y=polyPY, Q2XZero, Q2YZero;
//   printf("my id %d tx%d bx%d by%d bdx%d bdy%d gdx%d gdy%d\n", id, threadIdx.x, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
  if(id>maxID) return;

  // int sizePBlockX=(sizeP-1)/(gridDim.x*gridDim.y);
  // int sizePBlockY=((sizeP-1)%(gridDim.x*gridDim.y))/gridDim.x;
  // int maxID=(maxY*gridDim.x+maxX)*blockDim.x+(blockDim.x-1)+blockDim.x;

  point P1, P2, Q1, Q2;
  int pid=id;
  // int  phaseSize=sizeQ/MAX_POLY2_SIZE;
  Q2XZero=polyQX[0];
  Q2YZero=polyQY[0];
  if(id>=sizeP){
    poly1X=polyQX; 
    poly1Y=polyQY; 
    pid=id-sizeP;
    size2=sizeP;
    // phaseSize=sizeP/MAX_POLY2_SIZE;
    Q2XZero=polyPX[0];
    Q2YZero=polyPY[0];
  }
  int  phaseSize=(size2+MAX_POLY2_SIZE-1)/MAX_POLY2_SIZE;
  // if(id<(sizeP+sizeQ)){
    P1.x = poly1X[pid];
    P1.y = poly1Y[pid];
    //polygon1 is P and polygon2 is Q
    if(pid==id && pid==sizeP-1){
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    }else if(pid!=id && pid == sizeQ-1){ //polygon2 is P and polygon1 is Q
      P2.x = poly1X[0];
      P2.y = poly1Y[0];
      // printf("sp %d\n", pid);
    } else { //no need reset. Normal case
      P2.x = poly1X[pid+1];
      P2.y = poly1Y[pid+1];
    }
  // }
  // printf("%d \n", phaseSize);
  for(int phase=0; phase<phaseSize; phase++){
    // if(id==0)    printf("**++ %d %d %d %d \n", idx, pid, MAX_POLY2_SIZE, phase);
    // if(id<sizeP){
    size=MAX_POLY2_SIZE;
    if(phase==phaseSize-1 && size2%MAX_POLY2_SIZE!=0)size=size2%MAX_POLY2_SIZE;

    if(id<sizeP && ((phase==phaseSize-1 && idx<size) || phase!=phaseSize-1)){
        // load data into shared memory collaboratively
        poly2X_shared[idx]=polyQX[idx+(phase*MAX_POLY2_SIZE)];
        poly2Y_shared[idx]=polyQY[idx+(phase*MAX_POLY2_SIZE)];
        if(idx==MAX_POLY2_SIZE-1 && phase!=phaseSize-1){
          poly2X_shared[idx+1]=polyQX[idx+1+(phase*MAX_POLY2_SIZE)];
          poly2Y_shared[idx+1]=polyQY[idx+1+(phase*MAX_POLY2_SIZE)];
        }
        // printf("*4*++ %d %d %d %d \n", idx, pid, MAX_POLY2_SIZE, phase);
    } 
    else if(id>=sizeP && ((phase==phaseSize-1 && idx<(size)) || phase!=phaseSize-1)){
        // printf("**++ %d %d %d %d \n", idx, pid, MAX_POLY2_SIZE, phase);
        // load data into shared memory collaboratively
        poly2X_shared[idx]=polyPX[idx+(phase*MAX_POLY2_SIZE)];
        poly2Y_shared[idx]=polyPY[idx+(phase*MAX_POLY2_SIZE)];
        if(idx==MAX_POLY2_SIZE-1 && phase!=phaseSize-1){
          poly2X_shared[idx+1]=polyPX[idx+1+(phase*MAX_POLY2_SIZE)];
          poly2Y_shared[idx+1]=polyPY[idx+1+(phase*MAX_POLY2_SIZE)];
        }
        // printf("**++ %d %d \n", idx, MAX_POLY2_SIZE);
    }
        // printf("**++ %d %d %d %d \n", idx, pid, MAX_POLY2_SIZE, phase);
    __syncthreads();
    // if(id<(sizeP+sizeQ)){
    // if(id>=(sizeP)){
    // if(id<(sizeP)){
        // if(phase==phaseSize-1) size=size2%MAX_POLY2_SIZE;
        // else size=MAX_POLY2_SIZE;
        // printf("size %d\n", size);
        for(int qid=0; qid<size; qid++){   
            // if(checkCount!=qid+(phase*MAX_POLY2_SIZE)){
            // if(id==101242 && qid==0){
            //   printf("Wrong! I am id %d phase %d qid %d checkcount %d size %d\n", id, phase, qid, checkCount, size);}
            // checkCount++;
            Q1.x = poly2X_shared[qid];
            Q1.y = poly2Y_shared[qid];

            // reset P2 vertex of last edge to first vertex
            if(qid == size-1 && phase==phaseSize-1){
              Q2.x=Q2XZero;
              Q2.y=Q2YZero;
            }else{
              Q2.x=poly2X_shared[qid+1];
              Q2.y=poly2Y_shared[qid+1];
            }
            
            // determine intersection or overlap type
            int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
            if(i!=0){
              count1++;
              if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7)))
                  count2++;
            }
        }
    // }
    __syncthreads();
  }

  // if(id>=sizeP && id<sizeQ) printf("cs %d %d\n", count1, count2);
  // if(id==0) printf("sss %d %d\n", sizeP, sizeQ);
  count2++; //represent the parent vertex 
  if(id<sizeP){
    psP1[pid]=count1;
    psP2[pid]=count2;
  }else if(id>= sizeP){
  // printf("counts %d %d\n", count1, count2);
    psQ1[pid]=count1;
    psQ2[pid]=count2;
  }
  if(pid==0) printf("checkcount %d %d\n", id, checkCount);
  // __syncthreads();
  // thrust::exclusive_scan(thrust::device, psP1, psP1 + sizeP+1, psP1);   //sizeP location contains the total size of the count1
  // thrust::exclusive_scan(thrust::device, psP2, psP2 + sizeP+1, psP2);
  // thrust::exclusive_scan(thrust::device, psQ1, psQ1 + sizeQ+1, psQ1);   //sizeQ location contains the total size of the count1
  // thrust::exclusive_scan(thrust::device, psQ2, psQ2 + sizeQ+1, psQ2);
  // printf("id %d count1 %d count2  %d (%f,%f) (%f,%f)\n", id, count1, count2, P1.x, P1.y, P2.x, P2.y);
  // __syncthreads();
  // if(id==0){
  //   printf("%d \n", sizeP);
  //   for(int ii=0; ii<sizeP; ++ii){
  //     // printf("%d *%d ", psP1[ii], psP2[ii]);
  //     printf("%d ", psP1[ii]);
  //   }
  //   printf("\nend\n");
  //   for(int ii=0; ii<sizeQ; ++ii){
  //     // printf("%d *%d ", psQ1[ii], psQ2[ii]);
  //     printf("%d ", psQ1[ii]);
  //   }
  //   printf("\nend\n");
  // }
}*/

// simple bool check CMBR filter
__global__ void gpuCountIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeP2, int sizeQ,
                  int *psP1, int *psP2, int *boolPIndex, int *lsmf_count){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  int idx=threadIdx.x;  
  __shared__ double poly2X_shared[MAX_POLY2_SIZE+1], poly2Y_shared[MAX_POLY2_SIZE+1] /*+1 for halo next*/;
 double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=0, qid, count_lsmf=0;
  point P1, P2, Q1, Q2;

  int tiles=(sizeQ+MAX_POLY2_SIZE-1)/MAX_POLY2_SIZE;
  int tileCellsPerThread=MAX_POLY2_SIZE/blockDim.x;
  if(id<sizeP){
    P1.x = polyPX[id];
    P1.y = polyPY[id];
    P2.x = polyPX[(id+1)%sizeP];
    P2.y = polyPY[(id+1)%sizeP];
  }
  for(int tileId=0; tileId<tiles; tileId++){
    size=MAX_POLY2_SIZE;
    qid=idx*SHARED_MEMORY_PADDING;
    if(tileId==tiles-1 && sizeQ%MAX_POLY2_SIZE!=0){
      size=sizeQ%MAX_POLY2_SIZE;
      qid=0;
    }
    for(int localId=0; localId<tileCellsPerThread; ++localId){
      if(tileId!=tiles-1 || (tileId==tiles-1 && idx<size)){
        // load data into shared memory collaboratively
        poly2X_shared[idx+(blockDim.x*localId)]=polyQX[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)];
        poly2Y_shared[idx+(blockDim.x*localId)]=polyQY[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)];
        if(tileId!=tiles-1 && idx==blockDim.x-1 && localId==tileCellsPerThread-1){
          poly2X_shared[idx+(blockDim.x*localId)+1]=polyQX[idx+(blockDim.x*localId)+1+(tileId*MAX_POLY2_SIZE)];
          poly2Y_shared[idx+(blockDim.x*localId)+1]=polyQY[idx+(blockDim.x*localId)+1+(tileId*MAX_POLY2_SIZE)];
        }
      }
    } 
    __syncthreads();
    if(boolPIndex[id]) 
    {
      for(int qCount=0; qCount<size; qid=((qid+1)%size), ++qCount){   
      // for(int qid=0; qid<size; qid++){  
        Q1.x = poly2X_shared[qid];
        Q1.y = poly2Y_shared[qid];
        // reset P2 vertex of last edge to first vertex
        if(tileId==tiles-1 && qid==size-1){
          Q2.x=polyQX[0];
          Q2.y=polyQY[0];
        }else{
          Q2.x=poly2X_shared[qid+1];
          Q2.y=poly2Y_shared[qid+1];
        }      
        // if MBRs of two edges does not have a CMBR, there cannot be any intersection at all
        if(gpuLSMF(P1, P2, Q1, Q2))
        {
          count_lsmf++;
          // determine intersection or overlap type
          int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
          if(i!=0){
            count1++;
            if(i==1 || i==3 || i==5 || i==7)
              count2++;
          }        
        }
      } 
    }
    __syncthreads();
  }
  if(id<sizeP){
    count2++; //represent the parent vertex 
    psP1[id]=count1;
    psP2[id]=count2; 
    lsmf_count[id]=count_lsmf;
    // printf("%d %d\n", id, count_lsmf);
  }
}

// CMBR filter version
__global__ void gpuCountIntersections1(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeP2, int sizeQ,
                  int *psP1, int *psP2, int *boolPIndex){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  int idx=threadIdx.x;  
  __shared__ double poly2X_shared[MAX_POLY2_SIZE+1], poly2Y_shared[MAX_POLY2_SIZE+1] /*+1 for halo next*/;
 double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=0, qid;
  point P1, P2, Q1, Q2;

  int tiles=(sizeQ+MAX_POLY2_SIZE-1)/MAX_POLY2_SIZE;
  int tileCellsPerThread=MAX_POLY2_SIZE/blockDim.x;
  if(id<sizeP2){
    P1.x = polyPX[boolPIndex[id]];
    P1.y = polyPY[boolPIndex[id]];
    P2.x = polyPX[(boolPIndex[id]+1)%sizeP];
    P2.y = polyPY[(boolPIndex[id]+1)%sizeP];
  }
  for(int tileId=0; tileId<tiles; tileId++){
    size=MAX_POLY2_SIZE;
    qid=idx*SHARED_MEMORY_PADDING;
    if(tileId==tiles-1 && sizeQ%MAX_POLY2_SIZE!=0){
      size=sizeQ%MAX_POLY2_SIZE;
      qid=0;
    }
    for(int localId=0; localId<tileCellsPerThread; ++localId){
      if(tileId!=tiles-1 || (tileId==tiles-1 && idx<size)){
        // load data into shared memory collaboratively
        poly2X_shared[idx+(blockDim.x*localId)]=polyQX[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)];
        poly2Y_shared[idx+(blockDim.x*localId)]=polyQY[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)];
        if(tileId!=tiles-1 && idx==blockDim.x-1 && localId==tileCellsPerThread-1){
          poly2X_shared[idx+(blockDim.x*localId)+1]=polyQX[idx+(blockDim.x*localId)+1+(tileId*MAX_POLY2_SIZE)];
          poly2Y_shared[idx+(blockDim.x*localId)+1]=polyQY[idx+(blockDim.x*localId)+1+(tileId*MAX_POLY2_SIZE)];
        }
      }
    } 
    __syncthreads();
    for(int qCount=0; qCount<size; qid=((qid+1)%size), ++qCount){   
      Q1.x = poly2X_shared[qid];
      Q1.y = poly2Y_shared[qid];

      // reset P2 vertex of last edge to first vertex
      if(tileId==tiles-1 && qid==size-1){
        Q2.x=polyQX[0];
        Q2.y=polyQY[0];
      }else{
        Q2.x=poly2X_shared[qid+1];
        Q2.y=poly2Y_shared[qid+1];
      }
      // if MBRs of two edges does not have a CMBR, there cannot be any intersection at all
      if(gpuLSMF(P1, P2, Q1, Q2)){
        // determine intersection or overlap type
        int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
        // if(id==3)  printf("/sp 3// %d %d %d(%f %f) (%f %f) %d\n", id, tileId, qid, P1.x, P1.y, Q1.x, Q1.y, i);
        if(i!=0){
          // printf("/// %d %d (%f %f) (%f %f)\n", id, i, P1.x, P1.y, Q1.x, Q1.y);
          count1++;
          if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7)))
            count2++;
        }
      }
    } 
    __syncthreads();
  }
  if(id<sizeP2){
    count2++; //represent the parent vertex 
    psP1[boolPIndex[id]]=count1;
    psP2[boolPIndex[id]]=count2; 
  }
}

// with advanced CMBR filter
/*__global__ void gpuCountIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeP2, int sizeQ, int sizeQ2,
                  int *psP1, int *psP2, int *boolPIndex, int *boolQIndex){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  int idx=threadIdx.x;  
  __shared__ double poly2X_shared[MAX_POLY2_SIZE+1], poly2Y_shared[MAX_POLY2_SIZE+1] //+1 for halo next;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, size=0, qid;
  point P1, P2, Q1, Q2;
  int tmpCount=0;

  int tiles=(sizeQ2*2+MAX_POLY2_SIZE-1)/MAX_POLY2_SIZE;
  int tileCellsPerThread=MAX_POLY2_SIZE/blockDim.x;
  if(id==0) printf("^^ %d %d %d\n", sizeQ2, tiles, tileCellsPerThread);
  if(id<sizeP2){
    P1.x = polyPX[boolPIndex[id]];
    P1.y = polyPY[boolPIndex[id]];
    P2.x = polyPX[(boolPIndex[id]+1)%sizeP];
    P2.y = polyPY[(boolPIndex[id]+1)%sizeP];
  }
  for(int tileId=0; tileId<tiles; tileId++){
    size=MAX_POLY2_SIZE;
    qid=idx*SHARED_MEMORY_PADDING;
    if(tileId==tiles-1 && sizeQ2%MAX_POLY2_SIZE!=0){
      size=sizeQ2%MAX_POLY2_SIZE;
      qid=0;
    }
    for(int localId=0; localId<tileCellsPerThread; ++localId){
        // if(id==3) printf("val %d %f\n", (blockDim.x*localId), polyQY[(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)]);
      if(tileId!=tiles-1 || (tileId==tiles-1 && idx<size)){
        // load data into shared memory collaboratively
        
        // evn ID threads copy direct location data
        if(idx%2==0){
          poly2X_shared[idx+(blockDim.x*localId)]=polyQX[boolQIndex[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)]];
          poly2Y_shared[idx+(blockDim.x*localId)]=polyQY[boolQIndex[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)]];
        }else{
          poly2X_shared[idx+(blockDim.x*localId)]=polyQX[boolQIndex[(idx-1)+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)]+1];
          poly2Y_shared[idx+(blockDim.x*localId)]=polyQY[boolQIndex[(idx-1)+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)]+1];
        }
        
        
        if(tileId!=tiles-1 && idx==blockDim.x-1 && localId==tileCellsPerThread-1){
          if(idx%2==0){
            poly2X_shared[idx+(blockDim.x*localId)+1]=polyQX[(boolQIndex[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)+1])];
            poly2Y_shared[idx+(blockDim.x*localId)+1]=polyQY[(boolQIndex[idx+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)+1])];
          }else{
            poly2X_shared[idx+(blockDim.x*localId)+1]=polyQX[(boolQIndex[(idx-1)+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)]+1)];
            poly2Y_shared[idx+(blockDim.x*localId)+1]=polyQY[(boolQIndex[(idx-1)+(blockDim.x*localId)+(tileId*MAX_POLY2_SIZE)]+1)];
          }
        }
      }
    } 
    tmpCount+=size;
    __syncthreads();
    for(int qCount=0; qCount<size; qid=((qid+1)%size), ++qCount){   
    // for(int qid=0; qid<size; qid++){  
      Q1.x = poly2X_shared[qid];
      Q1.y = poly2Y_shared[qid];

      // reset P2 vertex of last edge to first vertex
      if(tileId==tiles-1 && qid==size-1){
        Q2.x=polyQX[(boolQIndex[0])];
        Q2.y=polyQY[(boolQIndex[0])];
      }else{
        Q2.x=poly2X_shared[qid+1];
        Q2.y=poly2Y_shared[qid+1];
      }
      // if(boolPIndex[id])
      {
        // if MBRs of two edges does not have a CMBR, there cannot be any intersection at all
        if(isCMBRBetweenEdges(P1, P2, Q1, Q2)){
          // determine intersection or overlap type
          int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
          // if(id==3)  printf("/sp 3// %d %d %d(%f %f) (%f %f) %d\n", id, tileId, qid, P1.x, P1.y, Q1.x, Q1.y, i);
          if(i!=0){
            // printf("/// %d %d (%f %f) (%f %f)\n", id, i, P1.x, P1.y, Q1.x, Q1.y);
            count1++;
            if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7)))
              count2++;
          }
        }
      }
    } 
    __syncthreads();
  }
  if(id==0) printf("count %d\n", tmpCount);
  if(id<sizeP2){
    count2++; //represent the parent vertex 
    psP1[boolPIndex[id]]=count1;
    psP2[boolPIndex[id]]=count2; 
  // printf("id %d %d %d \n", id, count1, count2);
  }
}*/

__global__ void gpuNeighborMap(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psQ1, int *psQ2,
                  int *neighborMapQ, int *boolQIndex, int *boolPIndex, int *lsmf_count, int *psf1_count, int *psf2_count){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, nonDegenCount=0, count_lsmf=0, count_psf1=0, count_psf2=0;

  if(id>=sizeQ) return;

  neighborMapQ[psQ2[id]+count2]=-100;  
  // check if the current edge has any intersections. If not return
  // printf("id %d %d %d \n", id, psQ1[id], psQ1[id+1]);
  // CMBR filter: check if the edge intersect with CMBR (from boolPIndex)
  // prefix sum filter: check if the current edge has any intersection count
  // if(boolQIndex[id])
  {
    if(psQ1[id+1]!=psQ1[id])
    {
      count_psf1++;
      point P1, P2, Q1, Q2;

      P1.x = polyQX[id];
      P1.y = polyQY[id];
      P2.x = polyQX[(id+1)%sizeQ];
      P2.y = polyQY[(id+1)%sizeQ];

      for(int qid=0; qid<sizeP; qid++){
        // if(boolPIndex[qid])
        {
          // prefix sum filter: check if the current edge has any intersection count      
          if(psP1[qid+1]!=psP1[qid])
          {
            count_psf2++;
            Q1.x = polyPX[qid];
            Q1.y = polyPY[qid];
            Q2.x = polyPX[(qid+1)%sizeP];
            Q2.y = polyPY[(qid+1)%sizeP];

            if(gpuLSMF(P1, P2, Q1, Q2))
            {
              count_lsmf++;
              // determine intersection or overlap type
              int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
              if(i!=0){
                count1++;
                if((id<sizeP && (i==1 || i==3 || i==5 || i==7)) || (id>=sizeP && (i==1 || i==3 || i==5 || i==7))){
                  nonDegenCount++;
                  count2=nonDegenCount;
                }
                else if((id<sizeP && (i==2 || i==4 || i==6 || i==8)) || (id>=sizeP && (i==2 || i==4 || i==6 || i==8)))
                  count2=0;

                neighborMapQ[psQ2[id]+count2]=qid;      
              }
            }
          }
        }
      }
    }
  }
  lsmf_count[id]=count_lsmf;
  psf1_count[id]=count_psf1;
  psf2_count[id]=count_psf2;
}

/*
-----------------------------------------------------------------
Function to calculate all intersections save them in the correct 
location using prefixsum arrays and make neighbor connections
Returns 
  *intersection arrays with orginal vertices in them x2 (P and Q)
  *neighbor arrays x2 (P and q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*//*
__global__ void gpuCalculateIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX, double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psP2, int *psQ1, int *psQ2, 
                  double *intersectionsP, double *intersectionsQ, double *intersectionsP2, double *intersectionsQ2,
                  int *alphaValuesP, int *alphaValuesQ, int *tmpBucketP, int *tmpBucketQ, int *alphaSortedIndiciesP, int *alphaSortedIndiciesQ,
                  int *neighborP, int *neighborQ, int *neighborP2, int *neighborQ2,
                  int *neighborMapQ,
                  int *initLabelsQ){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, nonDegenCount=0, size=sizeQ, indexIntP, indexIntQ, start, end, localI, neighborQId;
  double *poly1X=polyPX, *poly1Y=polyPY, *poly2X=polyQX, *poly2Y=polyQY;

  if(id>=sizeP+sizeQ) return;

  point P1, P2, Q1, Q2;
  int pid=id;
  if(id>=sizeP){
    size=sizeP;
    poly1X=polyQX; 
    poly1Y=polyQY; 
    poly2X=polyPX;
    poly2Y=polyPY;
    pid=id-sizeP;
    intersectionsQ[psQ2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsQ[psQ2[pid]*2+1]=poly1Y[pid];
    intersectionsQ2[psQ2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsQ2[psQ2[pid]*2+1]=poly1Y[pid];
    alphaValuesQ[psQ2[pid]]=-100;
    indexIntQ=getIntersectionStartIndex(pid, psQ1);
  } else {
    intersectionsP[psP2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsP[psP2[pid]*2+1]=poly1Y[pid];
    
    intersectionsP2[psP2[pid]*2]=poly1X[pid];       //consider edge for the intersection array
    intersectionsP2[psP2[pid]*2+1]=poly1Y[pid];
    alphaValuesP[psP2[pid]]=-100;
    indexIntP=getIntersectionStartIndex(pid, psP1);
  }

  P1.x = poly1X[pid];
  P1.y = poly1Y[pid];
  //polygon1 is P and polygon2 is Q
  if(pid==id && pid==sizeP-1){
    P2.x = poly1X[0];
    P2.y = poly1Y[0];
    // printf("sp %d\n", pid);
  }else if(pid!=id && pid == sizeQ-1){ //polygon2 is P and polygon1 is Q
    P2.x = poly1X[0];
    P2.y = poly1Y[0];
    // printf("sp %d\n", pid);
  } else { //no need reset. Normal case
    P2.x = poly1X[pid+1];
    P2.y = poly1Y[pid+1];
  }

  for(int qid=0; qid<size; qid++){
    // P1.x = poly1X[pid];
    // P1.y = poly1Y[pid];

    Q1.x = poly2X[qid];
    Q1.y = poly2Y[qid];
    Q2.x = poly2X[qid+1];
    Q2.y = poly2Y[qid+1];

    // reset P2 vertex of last edge to first vertex
    if(qid == size-1){
      Q2.x = poly2X[0];
      Q2.y = poly2Y[0];
    }
    // //polygon1 is P and polygon2 is Q
    // if(pid==id && pid==sizeP-1){
    //   P2.x = poly1X[0];
    //   P2.y = poly1Y[0];
    //   // printf("sp %d\n", pid);
    // }else if(pid!=id && pid == sizeQ-1){ //polygon2 is P and polygon1 is Q
    //   P2.x = poly1X[0];
    //   P2.y = poly1Y[0];
    //   // printf("sp %d\n", pid);
    // } else { //no need reset. Normal case
    //   P2.x = poly1X[pid+1];
    //   P2.y = poly1Y[pid+1];
    // }
    // determine intersection or overlap type
    int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
    if(i && id<sizeP){
      count1++;
      if(i==1 || i==3 || i==5 || i==7){
        nonDegenCount++;
        count2=nonDegenCount;
      }
      else if(i==2 || i==4 || i==6 || i==8)
        count2=0;
      // start=psQ2[neighborMapP[psP2[pid]+count2]];
      // end=psQ2[neighborMapP[psP2[pid]+count2]+1];
      start=psQ2[qid];
      end=psQ2[qid+1];

      if(i!=5){
        // local search to find the index of qid
        for(localI=start; localI<end; ++localI){
          if(pid==neighborMapQ[localI]){
            neighborQId=localI;
            neighborP[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
            neighborP2[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
            neighborQ[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
            neighborQ2[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
            localI=end+2; // break; 
          }
        }
      }else{
        neighborQId=start;
        neighborP[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
        neighborP2[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
        neighborQ[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
        neighborQ2[neighborQId]=psP2[pid]+count2+1;
        
        for(localI=start; localI<end; ++localI){
          if(pid==neighborMapQ[localI]){
            neighborQId=localI;
            neighborP[psP2[pid]]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
            neighborP2[psP2[pid]]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
            neighborQ[neighborQId]=psP2[pid]+1;   //+1 acting as a padding and helps to identify 0 being empty 
            neighborQ2[neighborQId]=psP2[pid]+1;   //+1 acting as a padding and helps to identify 0 being empty 
            localI=end+2; // break; 
          }
        }
      }
      switch(i) {
        // case X_INTERSECTION:
        // I and I
        case 1:
          I = add(mulScalar((1.0-alpha), P1), mulScalar(alpha, P2));
          intersectionsP[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsP[(psP2[pid]+count2)*2+1]=I.y;
          intersectionsP2[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsP2[(psP2[pid]+count2)*2+1]=I.y;
          alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // X-overlap
        // P1 and I(=P1 I is in Q)
        // I(=Q1 I is in P) and Q1
        case 5:
          intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
          intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
          alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case T_INTERSECTION_Q:
        // case T_OVERLAP_Q:
        // P1 and I(=P1 is in Q)
        case 2:
        case 6:
          alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
        break;
        // case T_INTERSECTION_P:
        // case T_OVERLAP_P:
        // I(=Q1 is in P) and Q1
        case 3:
        case 7:
          intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
          intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
          intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
          alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case V_INTERSECTION:
        // case V_OVERLAP:
        // P1 and Q1
        case 4:
        case 8:
          alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
      } 
    } else if(i && id>=sizeP){
      initLabelsQ[(psQ2[pid]+count2)]=-100;    //make init label to default -100 
      count1++;
      if(i==1 || i==3 || i==5 || i==7){
        nonDegenCount++;
        count2=nonDegenCount;
      }
      else if(i==2 || i==4 || i==6 || i==8)
        count2=0;        
      switch(i) {
        // case X_INTERSECTION:
        case 1:
          I = add(mulScalar((1.0-alpha), P1), mulScalar(alpha, P2));
          // I.x=getValueTolarence(I.x);
          // I.y=getValueTolarence(I.y);
          intersectionsQ[(psQ2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsQ[(psQ2[pid]+count2)*2+1]=I.y;
          intersectionsQ2[(psQ2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
          intersectionsQ2[(psQ2[pid]+count2)*2+1]=I.y;
          alphaValuesQ[psQ2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case X_OVERLAP:
        case 5:
          intersectionsQ[(psQ2[pid]+count2)*2]=Q1.x;    
          intersectionsQ[(psQ2[pid]+count2)*2+1]=Q1.y;
          intersectionsQ2[(psQ2[pid]+count2)*2]=Q1.x;    
          intersectionsQ2[(psQ2[pid]+count2)*2+1]=Q1.y;
          alphaValuesQ[psQ2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*beta;
          break;
        // case T_INTERSECTION_Q:
        // case T_OVERLAP_Q: 
        // was 2, 6
        case 3:
        case 7:
          intersectionsQ[(psQ2[pid]+count2)*2]=Q1.x;
          intersectionsQ[(psQ2[pid]+count2)*2+1]=Q1.y;
          intersectionsQ2[(psQ2[pid]+count2)*2]=Q1.x;
          intersectionsQ2[(psQ2[pid]+count2)*2+1]=Q1.y;
          alphaValuesQ[psQ2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
        // case T_INTERSECTION_P:
        // case T_OVERLAP_P:
        // was 3, 7
        case 2:
        case 6:
          alphaValuesQ[psQ2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
        break;
        // case V_INTERSECTION:
        // case V_OVERLAP:
        case 4:
        case 8:
          alphaValuesQ[psQ2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
          break;
      } 
    }
  }
  // --------------------------------------------------------------------------------------------
  // local sort for each edge, start to end
  // --------------------------------------------------------------------------------------------
  if(id<sizeP){
    int start=psP2[pid], end=psP2[pid+1];
    // sort intersection vertices in this edge locally
    if((end-start)>2){
      gpuRadixsort(alphaValuesP, tmpBucketP, alphaSortedIndiciesP, start+1, end);
      // using sorted index array, change intersection locations in the array and neighbors
      // decending order JUST FOR TESING
      // for(int i=start+1, j=end-1; i<end; ++i, j--){
      // acending order of alpha values 
      for(int i=start+1, j=start+1; i<end; i++, j++){
        alphaValuesP[i]=tmpBucketP[j];
        intersectionsP[i*2]=intersectionsP2[alphaSortedIndiciesP[j]*2];
        intersectionsP[i*2+1]=intersectionsP2[alphaSortedIndiciesP[j]*2+1];
        neighborP[i]=neighborP2[alphaSortedIndiciesP[j]];
        neighborQ[neighborP2[alphaSortedIndiciesP[j]]-1]=i+1; //+1 is the padding. When reading do -1
        neighborQ2[neighborP2[alphaSortedIndiciesP[j]]-1]=i+1; //updates neighborQ2 as the new original to be used with sorted Q array
      } 
    } 
  }
  // --------------------------------------------------------------------------------------------
}*/

__global__ void gpuCalculateIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX, double *polyQY, 
                  int sizeP, int sizeQ, 
                  int *psP1, int *psP2, int *psQ1, int *psQ2, 
                  double *intersectionsP, double *intersectionsQ, double *intersectionsP2, double *intersectionsQ2,
                  int *alphaValuesP, int *alphaValuesQ, int *tmpBucketP, int *alphaSortedIndiciesP,
                  int *neighborP, int *neighborQ, int *neighborP2, int *neighborQ2,
                  int *neighborMapQ, int *boolQIndex, int *boolPIndex, int *lsmf_count, int *psf1_count, int *psf2_count){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  double alpha;
  double beta;
  point I;
  int count1=0, count2=0, nonDegenCount=0, start, end, localI, neighborQId;
  int count_lsmf=0, count_psf1=0, count_psf2=0;

  if(id>=sizeP) return;

  point P1, P2, Q1, Q2;
  int pid=id;

  intersectionsP[psP2[pid]*2]=polyPX[pid];       //consider edge for the intersection array
  intersectionsP[psP2[pid]*2+1]=polyPY[pid];
  intersectionsP2[psP2[pid]*2]=polyPX[pid];       //consider edge for the intersection array
  intersectionsP2[psP2[pid]*2+1]=polyPY[pid];
  alphaValuesP[psP2[pid]]=-100;

  if(id<sizeQ){
    intersectionsQ[psQ2[pid]*2]=polyQX[pid];       //consider edge for the intersection array
    intersectionsQ[psQ2[pid]*2+1]=polyQY[pid];
    intersectionsQ2[psQ2[pid]*2]=polyQX[pid];       //consider edge for the intersection array
    intersectionsQ2[psQ2[pid]*2+1]=polyQY[pid];
  }

  // prefix sum filter: check if the current edge has any intersection count      
  if(psP1[id+1]!=psP1[id])
  // CMBR filter followed by prefix sum filter
  // if(boolPIndex[id] && psP1[id+1]!=psP1[id])
  {
    count_psf1++;
    P1.x = polyPX[pid];
    P1.y = polyPY[pid];
    P2.x = polyPX[(pid+1)%sizeP];
    P2.y = polyPY[(pid+1)%sizeP];

    for(int qid=0; qid<sizeQ; qid++){
      // prefix sum filter: check if the current edge has any intersection count      
      if(psQ1[qid+1]!=psQ1[qid])
      // CMBR filter followed by prefix sum filter
      // if(boolQIndex[qid] && psQ1[qid+1]!=psQ1[qid])
      {
        count_psf2++;
        Q1.x = polyQX[qid];
        Q1.y = polyQY[qid];
        Q2.x = polyQX[(qid+1)%sizeQ];
        Q2.y = polyQY[(qid+1)%sizeQ];

        if(gpuLSMF(P1, P2, Q1, Q2))
        {
          count_lsmf++;
          // determine intersection or overlap type
          int i = getIntersectType(P1, P2, Q1, Q2, alpha, beta);
          if(i){
            count1++;
            if(i==1 || i==3 || i==5 || i==7){
              nonDegenCount++;
              count2=nonDegenCount;
            }
            else if(i==2 || i==4 || i==6 || i==8)
              count2=0;
            start=psQ2[qid];
            end=psQ2[qid+1];

            if(i!=5){
              // local search to find the index of qid
              for(localI=start; localI<end; ++localI){
                if(pid==neighborMapQ[localI]){
                  neighborQId=localI;
                  neighborP[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborP2[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ2[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  localI=end+2; // break; 
                }
              }
            }else{
              neighborQId=start;
              neighborP[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
              neighborP2[psP2[pid]+count2]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
              neighborQ[neighborQId]=psP2[pid]+count2+1;   //+1 acting as a padding and helps to identify 0 being empty 
              neighborQ2[neighborQId]=psP2[pid]+count2+1;
              
              for(localI=start; localI<end; ++localI){
                if(pid==neighborMapQ[localI]){
                  neighborQId=localI;
                  neighborP[psP2[pid]]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborP2[psP2[pid]]=neighborQId+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ[neighborQId]=psP2[pid]+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  neighborQ2[neighborQId]=psP2[pid]+1;   //+1 acting as a padding and helps to identify 0 being empty 
                  localI=end+2; // break; 
                }
              }
            }
            switch(i) {
              // case X_INTERSECTION:
              // I and I
              case 1:
                I = add(mulScalar((1.0-alpha), P1), mulScalar(alpha, P2));
                intersectionsP[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
                intersectionsP[(psP2[pid]+count2)*2+1]=I.y;
                intersectionsP2[(psP2[pid]+count2)*2]=I.x;       //consider edge for the intersection array
                intersectionsP2[(psP2[pid]+count2)*2+1]=I.y;
                alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                intersectionsQ[neighborQId*2]=I.x;       //consider edge for the intersection array
                intersectionsQ[neighborQId*2+1]=I.y;
                intersectionsQ2[neighborQId*2]=I.x;       //consider edge for the intersection array
                intersectionsQ2[neighborQId*2+1]=I.y;
                alphaValuesQ[neighborQId]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
              // X-overlap
              // P1 and I(=P1 I is in Q)
              // I(=Q1 I is in P) and Q1
              case 5:
                intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
                intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
                alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                intersectionsQ[neighborQId*2]=P1.x;    
                intersectionsQ[neighborQId*2+1]=P1.y;
                intersectionsQ2[neighborQId*2]=P1.x;    
                intersectionsQ2[neighborQId*2+1]=P1.y;
                alphaValuesQ[neighborQId]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
              // case T_INTERSECTION_Q:
              // case T_OVERLAP_Q:
              // P1 and I(=P1 is in Q)
              case 2:
              case 6:
                alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                intersectionsQ[neighborQId*2]=P1.x;
                intersectionsQ[neighborQId*2+1]=P1.y;
                intersectionsQ2[neighborQId*2]=P1.x;
                intersectionsQ2[neighborQId*2+1]=P1.y;
                alphaValuesQ[neighborQId]=(int)pow(10, EPSILON_POSITIONS)*beta;
              break;
              // case T_INTERSECTION_P:
              // case T_OVERLAP_P:
              // I(=Q1 is in P) and Q1
              case 3:
              case 7:
                intersectionsP[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP[(psP2[pid]+count2)*2+1]=Q1.y;
                intersectionsP2[(psP2[pid]+count2)*2]=Q1.x;
                intersectionsP2[(psP2[pid]+count2)*2+1]=Q1.y;
                alphaValuesP[psP2[pid]+count2]=(int)pow(10, EPSILON_POSITIONS)*alpha;

                alphaValuesQ[psQ2[qid]]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
              // case V_INTERSECTION:
              // case V_OVERLAP:
              // P1 and Q1
              case 4:
              case 8:
                alphaValuesP[psP2[pid]]=(int)pow(10, EPSILON_POSITIONS)*alpha;
                
                alphaValuesQ[psQ2[qid]]=(int)pow(10, EPSILON_POSITIONS)*beta;
                break;
            } 
          } 
        }
      }
    }
  
    // --------------------------------------------------------------------------------------------
    // local sort for each edge, start to end
    // --------------------------------------------------------------------------------------------
    start=psP2[pid];
    end=psP2[pid+1];
    // sort intersection vertices in this edge locally
    if((end-start)>2){
      gpuRadixsort(alphaValuesP, tmpBucketP, alphaSortedIndiciesP, start+1, end);
      // using sorted index array, change intersection locations in the array and neighbors
      // decending order JUST FOR TESING
      // for(int i=start+1, j=end-1; i<end; ++i, j--){
      // acending order of alpha values 
      for(int i=start+1, j=start+1; i<end; i++, j++){
        alphaValuesP[i]=tmpBucketP[j];
        intersectionsP[i*2]=intersectionsP2[alphaSortedIndiciesP[j]*2];
        intersectionsP[i*2+1]=intersectionsP2[alphaSortedIndiciesP[j]*2+1];
        neighborP[i]=neighborP2[alphaSortedIndiciesP[j]];
        neighborQ[neighborP2[alphaSortedIndiciesP[j]]-1]=i+1; //+1 is the padding. When reading do -1
        neighborQ2[neighborP2[alphaSortedIndiciesP[j]]-1]=i+1; //updates neighborQ2 as the new original to be used with sorted Q array
      } 
    } 
  // --------------------------------------------------------------------------------------------
  }
  lsmf_count[id]=count_lsmf;
  psf1_count[id]=count_psf1;
  psf2_count[id]=count_psf2;
}

/*
-----------------------------------------------------------------
Function to save vertices of Q in edge wise sorted order
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuSortPolyQ(
                  int sizeQ, 
                  int *psQ2, 
                  double *intersectionsQ, double *intersectionsQ2,
                  int *alphaValuesQ, int *tmpBucketQ,  int *alphaSortedIndiciesQ,
                  int *neighborP, int *neighborQ, int *neighborQ2){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;

  if(id<sizeQ){
    int start=psQ2[id], end=psQ2[id+1];
    // sort intersection vertices in this edge locally
    if((end-start)>2){
      gpuRadixsort(alphaValuesQ, tmpBucketQ, alphaSortedIndiciesQ, start+1, end);
      // using sorted index array, change intersection locations in the array and neighbors
      // decending order JUST FOR TESING
      // for(int i=start+1, j=end-1; i<end; ++i, j--){
      // acending order of alpha values 
      for(int i=start+1, j=start+1; i<end; i++, j++){
        alphaValuesQ[i]=tmpBucketQ[j];////////////////?????????????????????? need to swap alpha too!!!
        // (x,y,alpha) tuple change in sorted order
        intersectionsQ[i*2]=intersectionsQ2[alphaSortedIndiciesQ[j]*2];
        intersectionsQ[i*2+1]=intersectionsQ2[alphaSortedIndiciesQ[j]*2+1];
        //neighbor array update
        neighborQ[i]=neighborQ2[alphaSortedIndiciesQ[j]];
        neighborP[neighborQ2[alphaSortedIndiciesQ[j]]-1]=i+1; //+1 is the padding. When reading do -1 //[]= i+1
      } 
    } 
  }
}

/*
-----------------------------------------------------------------
Function to calculate initial label
Returns 
  *initial labels x2 (P and Q)
Runs in GPU
Called from Host
-------------------------------------------------------------------
*/
__global__ void gpuCalculateInitLabel(
                int sizeP, int *psP2,
                double *intersectionsP, double *intersectionsQ, int *alphaValuesP, 
                int *neighborP,
                int sizeNP, int sizeNQ, int *initLabelsP, int *initLabelsQ){
  int id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  int pid=id;
  if(id>=sizeP) return;
  int start=psP2[pid], end=psP2[pid+1];
  // int start=psP2[id], end=psP2[id+1];
  int tmpId, nId, pMNId, pPNId;
  point pM, pP, qM, qP, current;
  int qMType, qPType, tmpIniLabel;
  int i;
  for(i=start; i<end; i++){
    initLabelsP[i]=-100;
    if(alphaValuesP[i]!=-100){    //consider intersections only
      current.x=intersectionsP[i*2]; 
      current.y=intersectionsP[i*2+1]; 
      tmpId=getCircularId(i-1, sizeNP);
      // determine local configuration at this intersection vertex
      pM.x=intersectionsP[tmpId*2];                // P-, predecessor of I on P
      pM.y=intersectionsP[tmpId*2+1];                // P-, predecessor of I on P
      // if(intersectionsP[tmpId*2+2]!=-100)
      if(alphaValuesP[tmpId]!=-100)
        pMNId=neighborP[tmpId]-1; //get neighbor id of P_m vertex
      else pMNId=-100;

      tmpId=getCircularId(i+1, sizeNP);
      pP.x=intersectionsP[tmpId*2];                // P+, successor of I on P
      pP.y=intersectionsP[tmpId*2+1];                // P+, successor of I on P
      if(alphaValuesP[tmpId]!=-100)
        pPNId=neighborP[tmpId]-1; //get neighbor id of P_p vertex
      else pPNId=-100;

      // nId=getNeighborIndex(i, neighborMapP, neighborQ);
      nId=neighborP[i]-1;
      tmpId=getCircularId(nId-1, sizeNQ);
      qM.x=intersectionsQ[tmpId*2];     // Q-, predecessor of I on Q
      qM.y=intersectionsQ[tmpId*2+1];     // Q-, predecessor of I on Q
      qMType=oracle(pMNId, pPNId, tmpId, qM, pM, current, pP);

      tmpId=getCircularId(nId+1, sizeNQ);
      qP.x=intersectionsQ[tmpId*2];     // Q+, successor of I on P
      qP.y=intersectionsQ[tmpId*2+1];     // Q+, successor of I on P
      qPType=oracle(pMNId, pPNId, tmpId, qP, pM, current, pP);

      tmpIniLabel=getInitialLabel(qMType, qPType);
      initLabelsP[i]=tmpIniLabel;
      initLabelsQ[nId]=tmpIniLabel;
    }
  }
}

/*
-----------------------------------------------------------------
Function to count how many intersection points and prefix sums
Returns 
  *count of non degenerate vertices x2 (P and Q)
  *intersection points with non degenrate vertices included x2
  *neighbor map x2
  *neighbor arrays x2
  *initial labels x2
Neighbor of a vertex (assume index i) in P can be read in O(1) time using
  neighborQ[neighborMapP[i]]
  for Q
    neighborP[neighborMapQ[i]]
Runs in CPU
Called from Host
-------------------------------------------------------------------
*/
void calculateIntersections(
                  double *polyPX, double *polyPY, 
                  double *polyQX,  double *polyQY, 
                  int sizeP, int sizeQ, double *cmbr,
                  int *countNonDegenIntP, int *countNonDegenIntQ, 
                  double **intersectionsP, double **intersectionsQ, int **alphaValuesP, int **alphaValuesQ,
                  int **initLabelsP, int **initLabelsQ,
                  int **neighborP, int **neighborQ){
    double *dev_polyPX, *dev_polyPY, *dev_polyQX, *dev_polyQY;
    int *dev_psP1, *dev_psP2, *dev_psQ1, *dev_psQ2, *dev_boolPsPX, *dev_boolPsQX, *dev_boolPX, *dev_boolQX;
    int psP1[sizeP+1], psP2[sizeP+1], psQ1[sizeQ+1], psQ2[sizeQ+1];
    int boolPsPX[sizeP+1], boolPsQX[sizeQ+1];
    hipEvent_t kernelStart0, kernelStart1, kernelStart2, kernelStart3, kernelStart4, kernelStart5, kernelStart6, kernelStart7, kernelStart8;
    hipEvent_t kernelStop0, kernelStop1, kernelStop2, kernelStop3, kernelStop4, kernelStop5, kernelStop6, kernelStop7, kernelStop8;
    int *dev_lsmf_count, lsmf_count[sizeP], *dev_psf1_count, psf1_count[sizeP], *dev_psf2_count, psf2_count[sizeP];
    int countCMBRP,countCMBRQ;

    // printf("cmbr %f %f %f %f\n",*(cmbr+0), *(cmbr+1), *(cmbr+2), *(cmbr+3));
    
    // Phase1: Count intersections in each block. Create prefix sums to find local locations in each thread 
    // Allocate memory in device 
    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart0);
        hipEventCreate(&kernelStop0);
    }
    hipMalloc((void **) &dev_polyPX, sizeP*sizeof(double));
    hipMalloc((void **) &dev_polyPY, sizeP*sizeof(double));
    hipMalloc((void **) &dev_polyQX, sizeQ*sizeof(double));
    hipMalloc((void **) &dev_polyQY, sizeQ*sizeof(double));
    hipMalloc((void **) &dev_psP1, (sizeP+1)*sizeof(int));
    hipMalloc((void **) &dev_psP2, (sizeP+1)*sizeof(int));
    hipMalloc((void **) &dev_psQ1, (sizeQ+1)*sizeof(int));
    hipMalloc((void **) &dev_psQ2, (sizeQ+1)*sizeof(int));

    hipMalloc((void **) &dev_boolPX, sizeP*sizeof(int));
    hipMalloc((void **) &dev_boolPsPX, (sizeP+1)*sizeof(int));
    hipMalloc((void **) &dev_boolQX, sizeQ*sizeof(int));
    hipMalloc((void **) &dev_boolPsQX, (sizeQ+1)*sizeof(int));

    hipMalloc((void **) &dev_lsmf_count, (sizeP)*sizeof(int));
    hipMalloc((void **) &dev_psf1_count, (sizeP)*sizeof(int));
    hipMalloc((void **) &dev_psf2_count, (sizeP)*sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_polyPX, polyPX, sizeP*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_polyPY, polyPY, sizeP*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_polyQX, polyQX, sizeQ*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_polyQY, polyQY, sizeQ*sizeof(double), hipMemcpyHostToDevice);

    int blocksPerGrid=((sizeP+sizeQ) + xThreadPerBlock - 1) / xThreadPerBlock;
    int xBlocksPerGrid=(blocksPerGrid + yBlockPerGrid - 1) / yBlockPerGrid;

    int blocksPerGridQ=(sizeQ + xThreadPerBlock - 1) / xThreadPerBlock;
    int xBlocksPerGridQ=(blocksPerGridQ + yBlockPerGrid - 1) / yBlockPerGrid;
    int blocksPerGridP=(sizeP + xThreadPerBlock - 1) / xThreadPerBlock;
    int xBlocksPerGridP=(blocksPerGridP + yBlockPerGrid - 1) / yBlockPerGrid;
    
    // ******size_t number_of_blocks = N/threads_per_block + (size_t)(N % threads_per_block != 0);
    dim3 dimBlock(xThreadPerBlock, yThreadPerBlock, 1);
    dim3 dimGridP(xBlocksPerGridP, yBlockPerGrid, 1); 
    dim3 dimGridQ(xBlocksPerGridQ, yBlockPerGrid, 1); 


    // CMBR filter 

    if(DEBUG_TIMING) hipEventRecord(kernelStart0);
    gpuCMBRFilter<<<dimGridP, dimBlock>>>(
                dev_polyPX, dev_polyPY, 
                cmbr[0], cmbr[1], cmbr[2], cmbr[3],
                sizeP, dev_boolPsPX, dev_psP1, dev_psP2);
    gpuCMBRFilter<<<dimGridQ, dimBlock>>>(
                dev_polyQX, dev_polyQY, 
                cmbr[0], cmbr[1], cmbr[2], cmbr[3],
                sizeQ, dev_boolPsQX, dev_psQ1, dev_psQ2);

    if(DEBUG_TIMING) hipEventRecord(kernelStop0);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop0);

    hipDeviceSynchronize();
  
    if(DEBUG_INFO_PRINT){
      hipMemcpy(&boolPsPX, dev_boolPsPX, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&boolPsQX, dev_boolPsQX, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);
      // count how many edges overlap with CMBRs
      countCMBRP=0;
      for(int x=0; x<sizeP; ++x) if(boolPsPX[x]) countCMBRP++;
      printf("\nP overlap count with CMBR %d ",countCMBRP);
      countCMBRQ=0;
      for(int x=0; x<sizeQ; ++x) if(boolPsQX[x]) countCMBRQ++;
      printf("Q overlap count with CMBR %d \n\n",countCMBRQ);
    }
/*
    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart7);
        hipEventCreate(&kernelStop7);
    }
    if(DEBUG_TIMING) hipEventRecord(kernelStart7);
    thrust::exclusive_scan(thrust::host, boolPsPX, boolPsPX + sizeP+1, boolPsPX);   
    thrust::exclusive_scan(thrust::host, boolPsQX, boolPsQX + sizeQ+1, boolPsQX);   
    if(DEBUG_TIMING) hipEventRecord(kernelStop7);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop7);

    hipDeviceSynchronize();

    // for(int ii=0; ii<sizeP; ++ii){
    //   if(boolPsPX[ii]!=ii) printf("*== %d\n", boolPsPX[ii]);
    // }
    // for(int ii=0; ii<sizeQ; ++ii){
    //   if(boolPsQX[ii]!=ii) printf("*== %d\n", boolPsQX[ii]);
    // }
    // printf("done\n");

    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart8);
        hipEventCreate(&kernelStop8);
    }
    hipMemcpy(dev_boolPsPX, boolPsPX, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_boolPsQX, boolPsQX, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);

    if(DEBUG_TIMING) hipEventRecord(kernelStart8);

    gpuSaveCMBRIntersectedIndicies<<<dimGridP, dimBlock>>>(
            dev_polyPX, dev_polyPY, 
            cmbr[0], cmbr[1], cmbr[2], cmbr[3],
            sizeP, dev_boolPX, dev_boolPsPX);

    gpuSaveCMBRIntersectedIndicies<<<dimGridQ, dimBlock>>>(
            dev_polyQX, dev_polyQY, 
            cmbr[0], cmbr[1], cmbr[2], cmbr[3],
            sizeQ, dev_boolQX, dev_boolPsQX);

    if(DEBUG_TIMING) hipEventRecord(kernelStop8);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop8);
    hipDeviceSynchronize();

    int sizeP2=boolPsPX[sizeP], sizeQ2=boolPsQX[sizeQ];
    printf("** %d %d\n", sizeP, sizeP2);
    printf("**-- %d %d\n", sizeQ, sizeQ2);

    int blocksPerGridP2=(sizeP2 + xThreadPerBlock - 1) / xThreadPerBlock;
    int blocksPerGridQ2=(sizeQ2 + xThreadPerBlock - 1) / xThreadPerBlock;
    int xBlocksPerGridP2=(blocksPerGridP2 + yBlockPerGrid - 1) / yBlockPerGrid;
    int xBlocksPerGridQ2=(blocksPerGridQ2 + yBlockPerGrid - 1) / yBlockPerGrid;
    dim3 dimGridP2(xBlocksPerGridP2, yBlockPerGrid, 1); 
    dim3 dimGridQ2(xBlocksPerGridQ2, yBlockPerGrid, 1); 
*/

    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart1);
        hipEventCreate(&kernelStop1);
    }
/*
    if(DEBUG_TIMING) hipEventRecord(kernelStart1);
      gpuCountIntersections<<<dimGridQ, dimBlock>>>(
      // gpuCountIntersections<<<dimGridQ2, dimBlock>>>(
            dev_polyQX, dev_polyQY, 
            dev_polyPX, dev_polyPY, 
            // sizeQ, sizeQ2, sizeP,
            sizeQ, 10, sizeP,
            // dev_psQ1, dev_psQ2, dev_boolQX);
            dev_psQ1, dev_psQ2, dev_boolPsQX);
      
      gpuCountIntersections<<<dimGridP, dimBlock>>>(
      // gpuCountIntersections<<<dimGridP2, dimBlock>>>(
            dev_polyPX, dev_polyPY, 
            dev_polyQX, dev_polyQY, 
            // sizeP, sizeP2, sizeQ,
            sizeP, 10, sizeQ,
            // dev_psP1, dev_psP2, dev_boolPX);
            dev_psP1, dev_psP2, dev_boolPsPX);

    if(DEBUG_TIMING) hipEventRecord(kernelStop1);
*/

    if(DEBUG_TIMING) hipEventRecord(kernelStart1);
    gpuCountIntersections<<<dimGridQ, dimBlock>>>(
    // gpuCountIntersections<<<dimGridQ2, dimBlock>>>(
          dev_polyQX, dev_polyQY, 
          dev_polyPX, dev_polyPY, 
          // sizeQ, sizeQ2, sizeP,
          sizeQ, 10, sizeP,
          // dev_psQ1, dev_psQ2, dev_boolQX);
          dev_psQ1, dev_psQ2, dev_boolPsQX, dev_lsmf_count);
    
    if(DEBUG_INFO_PRINT){
      hipDeviceSynchronize();
      hipMemcpy(&lsmf_count, dev_lsmf_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
      long sum=0;
      for(int xx=0; xx<sizeQ; ++xx){
        sum+=lsmf_count[xx];
        // if (lsmf_count[xx]!=0) printf("%d %d\n", xx, lsmf_count[xx]);
      }
      printf("Q lsmf count %d ", sum);
      printf("Q Avg lsmf count %f\n",(double)sum/sizeQ);
    }


    gpuCountIntersections<<<dimGridP, dimBlock>>>(
    // gpuCountIntersections<<<dimGridP2, dimBlock>>>(
          dev_polyPX, dev_polyPY, 
          dev_polyQX, dev_polyQY, 
          // sizeP, sizeP2, sizeQ,
          sizeP, 10, sizeQ,
          // dev_psP1, dev_psP2, dev_boolPX);
          dev_psP1, dev_psP2, dev_boolPsPX, dev_lsmf_count);
    
    if(DEBUG_INFO_PRINT){
      hipMemcpy(&lsmf_count, dev_lsmf_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
      long sum=0;
      for(int xx=0; xx<sizeP; ++xx){
        sum+=lsmf_count[xx];
      }
      printf("P lsmf count %d ", sum);
      printf("P Avg lsmf count %f\n",(double)sum/sizeP);
    }

    if(DEBUG_TIMING) hipEventRecord(kernelStop1);

    // int maxX=(sizeP+sizeQ-2)/(dimGrid.x*dimGrid.y);
    // int maxY=((sizeP+sizeQ-2)%(dimGrid.x*dimGrid.y))/dimGrid.x;
    // int maxID=(maxY*dimGrid.x+maxX)*dimBlock.x+(dimBlock.x-1)+dimBlock.x;

    dim3 dimGrid2(xBlocksPerGrid, yBlockPerGrid, 1);
    // if(DEBUG_TIMING) hipEventRecord(kernelStart1);
    // gpuCountIntersections1<<<dimGrid2, dimBlock>>>(
    //         dev_polyPX, dev_polyPY, 
    //         dev_polyQX, dev_polyQY, 
    //         sizeP, sizeQ,
    //         dev_psP1, dev_psP2, dev_psQ1, dev_psQ2);
    // if(DEBUG_TIMING) hipEventRecord(kernelStop1);

    hipMemcpy(&psP1, dev_psP1, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psP2, dev_psP2, (sizeP+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psQ1, dev_psQ1, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psQ2, dev_psQ2, (sizeQ+1)*sizeof(int), hipMemcpyDeviceToHost);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop1);
    hipDeviceSynchronize();

    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart2);
        hipEventCreate(&kernelStop2);
    }
    if(DEBUG_TIMING) hipEventRecord(kernelStart2);
    thrust::exclusive_scan(thrust::host, psP1, psP1 + sizeP+1, psP1);   //sizeP location contains the total size of the count1
    thrust::exclusive_scan(thrust::host, psP2, psP2 + sizeP+1, psP2);
    thrust::exclusive_scan(thrust::host, psQ1, psQ1 + sizeQ+1, psQ1);   //sizeQ location contains the total size of the count1
    thrust::exclusive_scan(thrust::host, psQ2, psQ2 + sizeQ+1, psQ2);
    if(DEBUG_TIMING) hipEventRecord(kernelStop2);

    if(DEBUG_TIMING) hipEventSynchronize(kernelStop2);

    // for (int i = 0; i < sizeQ+1; ++i){
    // for (int i = 0; i < 15+1; ++i){
    //   printf(" %d-%d ", i, psP2[i]);
    // }
    // printf("--- \n");

    // // for (int i = 0; i < sizeQ+1; ++i){
    // for (int i = 0; i < 15+1; ++i){
    //   printf(" %d-%d ", i, psQ2[i]);
    // }
    // printf("--- \n");
    hipDeviceSynchronize();

    //Phase2: NEW- Fill neighborMap
    int *dev_neighborMapQ;
    int *neighborMapQ;
    *countNonDegenIntP=psP2[sizeP];
    *countNonDegenIntQ=psQ2[sizeQ];

    if(DEBUG_INFO_PRINT){
      printf("Non-degen count P %d *****--- Q %d\n", *countNonDegenIntP-sizeP, *countNonDegenIntQ-sizeQ);
      printf("Intersection count P %d *****--- Q %d\n", psP1[sizeP], psQ1[sizeQ]);
    }

    dim3 dimGrid(xBlocksPerGrid, yBlockPerGrid, 1);
    // printf("blockDim %d gridDimx %d gridDimy %d\n", dimBlock.x, dimGrid.x, dimGrid.y);

    neighborMapQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));

    hipMalloc((void **) &dev_neighborMapQ, *countNonDegenIntQ*sizeof(int));

    if(DEBUG_TIMING){
        hipEventCreate(&kernelStart3);
        hipEventCreate(&kernelStop3);
    }
    hipMemcpy(dev_psP1, psP1, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_psP2, psP2, (sizeP+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_psQ1, psQ1, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_psQ2, psQ2, (sizeQ+1)*sizeof(int), hipMemcpyHostToDevice);

    if(DEBUG_TIMING) hipEventRecord(kernelStart3);
    // gpuNeighborMap1<<<dimGrid, dimBlock>>>(
    //         dev_polyPX, dev_polyPY, 
    //         dev_polyQX, dev_polyQY, 
    //         sizeP, sizeQ,  
    //         dev_psP2, dev_psQ2,
    //         dev_neighborMapP, dev_neighborMapQ);
    gpuNeighborMap<<<dimGridQ, dimBlock>>>(
            dev_polyPX, dev_polyPY, 
            dev_polyQX, dev_polyQY, 
            sizeP, sizeQ,  
            dev_psP1, dev_psQ1, dev_psQ2,
            dev_neighborMapQ, dev_boolPsQX, dev_boolPsPX, dev_lsmf_count, dev_psf1_count, dev_psf2_count);
    if(DEBUG_TIMING) hipEventRecord(kernelStop3);
  
    long int sum2, sum1;
    if(DEBUG_INFO_PRINT){
      hipMemcpy(&lsmf_count, dev_lsmf_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&psf1_count, dev_psf1_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(&psf2_count, dev_psf2_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
      long sum=0;
      sum2=0, sum1=0;
      for(int xx=0; xx<sizeQ; ++xx){
        sum+=lsmf_count[xx];
        sum1+=psf1_count[xx];
        sum2+=(unsigned long)psf2_count[xx];
        // printf("%d %d\n",psf1_count[xx], psf2_count[xx]);
      }
      sum2=81497*81497;
      printf("\nQ psf1 count %d ", sum1);
      printf("P psf2 count %d ", sum2);
      printf("lsmf count %d \n", sum);
      // printf("P Avg lsmf count %f\n",(double)sum/sizeP);
    }
// -----------------------------------------------------------------------------------------------------
  // remove after kernel testing
  // hipMemcpy(*neighborMapP, dev_neighborMapP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(*neighborMapQ, dev_neighborMapQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
// -----------------------------------------------------------------------------------------------------
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop3);

  // Phase 3: Calcualte intersections and save them in the arrays. Make neighbor connections
  int countIntersections=psP1[sizeP];

  int *alphaSortedIndiciesP, *alphaSortedIndiciesQ;
  double *dev_intersectionsP, *dev_intersectionsQ, *dev_intersectionsP2, *dev_intersectionsQ2;
  int *dev_neighborP, *dev_neighborQ, *dev_neighborP2, *dev_neighborQ2;
  int *dev_initLabelsP, *dev_initLabelsQ;
  int *dev_alphaValuesP, *dev_alphaValuesQ, *dev_tmpBucketP, *dev_tmpBucketQ, *dev_alphaSortedIndiciesP, *dev_alphaSortedIndiciesQ;

  *intersectionsP=(double *)malloc(*countNonDegenIntP*2*sizeof(double));
  *intersectionsQ=(double *)malloc(*countNonDegenIntQ*2*sizeof(double));
  *alphaValuesP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *alphaValuesQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  alphaSortedIndiciesP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  alphaSortedIndiciesQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  *initLabelsP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *initLabelsQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));
  *neighborP=(int *)malloc(*countNonDegenIntP*sizeof(int));
  *neighborQ=(int *)malloc(*countNonDegenIntQ*sizeof(int));

  for(int i=0; i<*countNonDegenIntQ; ++i){
    *(*initLabelsQ+i)=-100;
    *(*alphaValuesQ+i)=-100;
  }

  hipDeviceSynchronize();

  // Allocate memory in device 
  hipMalloc((void **) &dev_intersectionsP, *countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsP2, *countNonDegenIntP*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ, *countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_intersectionsQ2, *countNonDegenIntQ*2*sizeof(double));
  hipMalloc((void **) &dev_alphaValuesP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaValuesQ, *countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_tmpBucketP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_tmpBucketQ, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_alphaSortedIndiciesQ, *countNonDegenIntQ*sizeof(int));

  hipMalloc((void **) &dev_neighborP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborP2, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_neighborQ, *countNonDegenIntQ*sizeof(int));
  hipMalloc((void **) &dev_neighborQ2, *countNonDegenIntQ*sizeof(int));
  // hipMalloc((void **) &dev_neighborMapP2, *countNonDegenIntP*sizeof(int));
  // hipMalloc((void **) &dev_neighborMapQ2, *countNonDegenIntQ*sizeof(int));

  hipMemcpy(dev_alphaValuesQ, *alphaValuesQ, *countNonDegenIntQ*sizeof(int), hipMemcpyHostToDevice);

  
  if(DEBUG_TIMING){
    hipEventCreate(&kernelStart4);
    hipEventCreate(&kernelStop4);
  }

  if(DEBUG_TIMING) hipEventRecord(kernelStart4);
  gpuCalculateIntersections<<<dimGridP, dimBlock>>>(
        dev_polyPX, dev_polyPY, 
        dev_polyQX, dev_polyQY, 
        sizeP, sizeQ, 
        dev_psP1, dev_psP2, dev_psQ1, dev_psQ2, 
        dev_intersectionsP, dev_intersectionsQ, dev_intersectionsP2, dev_intersectionsQ2,
        dev_alphaValuesP, dev_alphaValuesQ, dev_tmpBucketP, dev_alphaSortedIndiciesP,
        dev_neighborP, dev_neighborQ, dev_neighborP2, dev_neighborQ2,
        dev_neighborMapQ, dev_boolPsQX, dev_boolPsPX, dev_lsmf_count, dev_psf1_count, dev_psf2_count);
  if(DEBUG_TIMING) hipEventRecord(kernelStop4);
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop4);

  hipDeviceSynchronize();


  if(DEBUG_INFO_PRINT){
    hipMemcpy(&lsmf_count, dev_lsmf_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psf1_count, dev_psf1_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&psf2_count, dev_psf2_count, (sizeP)*sizeof(int), hipMemcpyDeviceToHost);
    long sum=0;
    long sum1=0, sum2=0;
    for(int xx=0; xx<sizeP; ++xx){
      sum+=lsmf_count[xx];
      sum1+=psf1_count[xx];
      sum2+=psf2_count[xx];
    }
    printf("\nP psf1 count %d ", sum1);
    printf("Q psf2 count %d ", sum2);
    printf("lsmf count %d \n", sum);
    // printf("P Avg lsmf count %f\n",(double)sum/sizeP);
  }


  if(DEBUG_TIMING){
    hipEventCreate(&kernelStart5);
    hipEventCreate(&kernelStop5);
  }
  if(DEBUG_TIMING) hipEventRecord(kernelStart5);
  gpuSortPolyQ<<<dimGridQ, dimBlock>>>(
        sizeQ, 
        dev_psQ2, 
        dev_intersectionsQ, dev_intersectionsQ2,
        dev_alphaValuesQ, dev_tmpBucketQ,  dev_alphaSortedIndiciesQ,
        dev_neighborP, dev_neighborQ, dev_neighborQ2);
  if(DEBUG_TIMING) hipEventRecord(kernelStop5);
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop5);

  hipDeviceSynchronize();

  // Phase4: Inital label classificaiton
  // hipMemcpy(*initLabelsQ, dev_initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMalloc((void **) &dev_initLabelsP, *countNonDegenIntP*sizeof(int));
  hipMalloc((void **) &dev_initLabelsQ, *countNonDegenIntQ*sizeof(int));

  hipMemcpy(dev_initLabelsQ, *initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyHostToDevice);
 
  // negative alpha values are not handled explicitly since they are original vertices
  // ******No need to copy alpha values since they are only used to sort edge wise******
  // hipMemcpy(alphaSortedIndicies, dev_alphaSortedIndicies, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  if(DEBUG_TIMING){
    hipEventCreate(&kernelStart6);
    hipEventCreate(&kernelStop6);
  }

  if(DEBUG_TIMING) hipEventRecord(kernelStart6);
  gpuCalculateInitLabel<<<dimGridP, dimBlock>>>(
      sizeP,  dev_psP2,
      dev_intersectionsP, dev_intersectionsQ, dev_alphaValuesP,
      dev_neighborP,
      *countNonDegenIntP, *countNonDegenIntQ, dev_initLabelsP, dev_initLabelsQ);
  if(DEBUG_TIMING) hipEventRecord(kernelStop6);

  hipMemcpy(*intersectionsP, dev_intersectionsP, *countNonDegenIntP*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*intersectionsQ, dev_intersectionsQ, *countNonDegenIntQ*2*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborP, dev_neighborP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*neighborQ, dev_neighborQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(*neighborMapP, dev_neighborMapP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(*neighborMapQ, dev_neighborMapQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);

  hipMemcpy(*initLabelsP, dev_initLabelsP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*initLabelsQ, dev_initLabelsQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesP, dev_alphaValuesP, *countNonDegenIntP*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(*alphaValuesQ, dev_alphaValuesQ, *countNonDegenIntQ*sizeof(int), hipMemcpyDeviceToHost);
  
  if(DEBUG_TIMING) hipEventSynchronize(kernelStop6);
  
  hipDeviceSynchronize();

  float kernelTiming1=0, kernelTiming2=0, kernelTiming3=0, kernelTiming4=0, kernelTiming5=0, kernelTiming6=0;
  if(DEBUG_TIMING){
    hipEventElapsedTime(&kernelTiming1, kernelStart1, kernelStop1);
    hipEventElapsedTime(&kernelTiming2, kernelStart2, kernelStop2);
    hipEventElapsedTime(&kernelTiming3, kernelStart3, kernelStop3);
    hipEventElapsedTime(&kernelTiming4, kernelStart4, kernelStop4);
    hipEventElapsedTime(&kernelTiming5, kernelStart5, kernelStop5);
    hipEventElapsedTime(&kernelTiming6, kernelStart6, kernelStop6);
    printf("\ngpuCountIntersections kernel exe time(ms) %f\n", kernelTiming1);
    printf("prefixsum kernels exe time(ms) %f\n", kernelTiming2);
    printf("gpuNeighborMap kernel exe time(ms) %f\n", kernelTiming3);
    printf("gpuCalculateIntersections kernel exe time(ms) %f\n", kernelTiming4);
    printf("gpuSortPolyQ kernel exe time(ms) %f\n", kernelTiming5);
    printf("gpuCalculateInitLabel kernel exe time(ms) %f\n\n", kernelTiming6);
  }

  // int limitP=*countNonDegenIntP;
  // int limitQ=*countNonDegenIntQ;
  int limitP=10;
  int limitQ=10;

  // printf("intersectionP");
  // for (int i = 0; i < limitP*2; ++i){
  //   if(i%2==0) 
  //     printf("\n%d %d ", i/2, *(*alphaValuesP+(i/2)));
  //   // printf(" %f ", intersectionsP[i]);
  //   printf(" %f ", *(*intersectionsP+i));
  // }
  // printf("\n\nintersectionQ");
  // for (int i = 0; i < limitQ*2; ++i){
  //   if(i%2==0)
  //     printf("\n%d %d ", i/2, *(*alphaValuesQ+(i/2)));
  //   printf(" %f ", *(*intersectionsQ+i));
  // }
  // printf("\n\nalpha P\n");
  // for (int i = 0; i < *countNonDegenIntP; ++i){
  //   printf(" %d>%d ", i, alphaValuesP[i]);
  // }
  // printf("\n\nalpha Q\n");
  // for (int i = 0; i < *countNonDegenIntQ; ++i){
  //   printf(" %d>%d ", i, alphaValuesQ[i]);
  // }
  // printf("\n");
  // printf("\nneighbor P\n");
  // // for (int i = 0; i < limitP; ++i){
  // for (int i = 679; i < 682; ++i){
  //   printf(" %d-%d ", i, *(*neighborP+i));
  // }
  // printf("\nnneighbor Q\n");
  // for (int i = 0; i < limitQ; ++i){
  //   printf(" %d-%d ", i, *(*neighborQ+i));
  // }
  // printf("\n");
  // for (int i = 0; i < *countNonDegenIntP; ++i)
  // {
  //   printf(" %d-%d ", i, *(*neighborMapP+i));
  // }
  // printf("\n");
  // for (int i = 0; i < *countNonDegenIntQ; ++i)
  // {
  //   printf(" %d-%d ", i, *(*neighborMapQ+i));
  // }
  // printf("\nLabel P\n");
  // for (int i = 0; i < limitP; ++i){
  //   printf(" %d>%d ", i, *(*initLabelsP+i));
  // }
  // printf("\nLabel Q\n");
  // for (int i = 0; i < limitQ; ++i){
  //   printf(" %d>%d ", i, *(*initLabelsQ+i));
  // }
  // printf("\n");


  hipFree(dev_polyPX);
  hipFree(dev_polyPY);
  hipFree(dev_polyQX);
  hipFree(dev_polyQY);
}